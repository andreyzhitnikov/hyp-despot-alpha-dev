#include "hip/hip_runtime.h"
#include "GPU_LowerBoundPolicy.h"

#include <despot/GPUcore/GPUhistory.h>
#include <despot/GPUrandom_streams.h>
#include <despot/GPUutil/GPUcoord.h>
#include "GPU_Car_Drive.h"
using archaeopteryx::util::numeric_limits;
using despot::Dvc_History;
using despot::Dvc_RandomStreams;
using despot::Dvc_State;

#include <despot/GPUinterface/GPUpomdp.h>

DEVICE int Dvc_PedPomdpDoNothingPolicy::Action(
		int scenarioID, const Dvc_State* particles,
				Dvc_RandomStreams& streams,
				Dvc_History& history)
{

	return 2;
	//For despot with alpha vector update action cannot depend on observation
	//const Dvc_PomdpState &state=static_cast<const Dvc_PomdpState&>(particles[0]);
	//float carvel = state.car.vel;
	//if (carvel >= 0.001) return 2;
	//return -1;
}
DEVICE int Dvc_PedPomdpSmartPolicy::Action(
		int scenarioID, const Dvc_State* particles,
				Dvc_RandomStreams& streams,
				Dvc_History& history)
{

	const Dvc_PomdpState &state=static_cast<const Dvc_PomdpState&>(particles[0]);
	__shared__ int mindist[32];
	auto& carpos = path->way_points_[state.car.pos];


	float carvel = state.car.vel;

	mindist[threadIdx.x]=__float_as_int(numeric_limits<float>::infinity());
	__syncthreads();
	if (threadIdx.y<state.num) {
		auto& p = state.peds[threadIdx.y];
		bool infront=false;

		if(Dvc_ModelParams::IN_FRONT_ANGLE_DEG >= 180.0) {
			// inFront check is disabled in this case
			infront=true;
		}
		else
		{
			const Dvc_COORD& car_pos = path->way_points_[state.car.pos];
			const Dvc_COORD& forward_pos = path->way_points_[path->forward(state.car.pos, 1.0)];
			


			float d0 = Dvc_COORD::EuclideanDistance(car_pos, p.pos);


			if(d0 <= /*0.7*/3.5)
				infront=true;
			else
			{
				float d1 = Dvc_COORD::EuclideanDistance(car_pos, forward_pos);
				if(d1<=0)
					infront=true;
				else
				{
					float dot = Dvc_Vector::DotProduct(forward_pos.x - car_pos.x, forward_pos.y - car_pos.y,
							p.pos.x - car_pos.x, p.pos.y - car_pos.y);
					float cosa = dot / (d0 * d1);
					if(cosa > 1.0 + 1E-6 || cosa < -1.0 - 1E-6)
					{
						;
					
					}
					infront=cosa > in_front_angle_cos;
				}
			}
		}



		if(infront) {
			float d = Dvc_COORD::EuclideanDistance(carpos, p.pos);
			atomicMin(mindist+threadIdx.x, __float_as_int(d));
		}
	}
	__syncthreads();

	// TODO set as a param
	if (__int_as_float(mindist[threadIdx.x]) < /*2*/3.5) {
		return (carvel <= 0.01) ? 0 : 2;
	}

	if (__int_as_float(mindist[threadIdx.x]) < /*4*/5) {
		if (carvel > 1.0+1e-4) return 2;
		else if (carvel < 0.5-1e-4) return 1;
		else return 0;
	}


	return carvel >= Dvc_ModelParams::VEL_MAX-1e-4 ? 0 : 1;
	return 0;
}
