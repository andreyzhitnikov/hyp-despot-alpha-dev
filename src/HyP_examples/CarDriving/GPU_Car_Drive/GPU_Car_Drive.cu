#include "hip/hip_runtime.h"
#include "GPU_Car_Drive.h"
#include <despot/GPUcore/thread_globals.h>

#include <ped_pomdp.h>
#include <despot/util/coord.h>
#include <hip/driver_types.h>
#include <stddef.h>
#include "despot/GPUutil/GPUmemorypool.h"
#include "despot/GPUutil/GPUrandom.h"

#include "GPU_CarUpperBound.h"

#define THREADDIM 128
using namespace std;

using namespace despot;
using namespace Globals;


static GPU_MemoryPool<Dvc_PomdpState>* gpu_mainstate_pool_=NULL;
static GPU_MemoryPool<Dvc_PedStruct>* gpu_ped_pool_=NULL;

static Dvc_PedStruct **Dvc_tempPeds=NULL;
static Dvc_PedStruct **Hst_tempPeds=NULL;

static float** Dvc_temp_weight=NULL;
static int** Hst_temp_IDs=NULL;
static float** Hst_temp_weight=NULL;
static Dvc_PomdpState** Hst_temp_mainstates=NULL;
static Dvc_PomdpState* Managed_rootnode_particles=NULL;


DEVICE Dvc_Path* path=NULL;
DEVICE Dvc_COORD* goals=NULL;
DEVICE double freq=0;
DEVICE double in_front_angle_cos=0;

using namespace despot;
/* ==============================================================================
 * Dvc_PomdpState class
 * ==============================================================================*/

DEVICE Dvc_PomdpState::Dvc_PomdpState():num(0), peds(NULL)
{
}

DEVICE Dvc_PomdpState::Dvc_PomdpState(const Dvc_PomdpState& src)
{
	*this=src;
}


/**
 * CopyPeds_to_Particles kernel:
 * Copy pedestrian states in a combined source list (in contigeous memory) to destination particles
 * This is for copying back to CPU
 */

__global__ void CopyPeds_to_Particles(Dvc_PomdpState* dvc_particles,  Dvc_PedStruct* src)
{
	int scenarioID=blockIdx.x;
	int ped_id=threadIdx.x;

	Dvc_PomdpState* Dvc_i=dvc_particles+scenarioID;
	Dvc_PedStruct* Src_i=src+scenarioID*Dvc_ModelParams::N_PED_IN;

	if(ped_id<Dvc_i->num)
		Dvc_i->peds[ped_id]=Src_i[ped_id];



}


/**
 * CopyPeds_to_list kernel:
 * Copy pedestrian states in particles into a combined contigoues memory list
 * This is for copying back to CPU
 */

__global__ void CopyPeds_to_list(const Dvc_PomdpState* particles,  Dvc_PedStruct* peds_list)
{
	int scenarioID=blockIdx.x;
	int ped_id=threadIdx.x;
	const Dvc_PomdpState* Dvc_i = particles + scenarioID;
	Dvc_PedStruct* Des_i = peds_list + scenarioID * Dvc_ModelParams::N_PED_IN;

	if(ped_id<Dvc_i->num)
		Des_i[ped_id]=Dvc_i->peds[ped_id];
}

HOST void Dvc_PomdpState::CopyMainStateToGPU(Dvc_PomdpState* dvc_particles, int scenarioID, const PomdpState* hst_particle)
{
	dvc_particles[scenarioID].car.dist_travelled=hst_particle->car.dist_travelled;
	dvc_particles[scenarioID].car.pos=hst_particle->car.pos;
	dvc_particles[scenarioID].car.vel=hst_particle->car.vel;
	dvc_particles[scenarioID].num=hst_particle->num;
	dvc_particles[scenarioID].weight=hst_particle->weight;
	dvc_particles[scenarioID].state_id=hst_particle->state_id;
	dvc_particles[scenarioID].scenario_id=hst_particle->scenario_id;


	int Data_block_size=ModelParams::N_PED_IN;

	if(Globals::config.use_multi_thread_ && StreamManager::MANAGER.cuda_streams)
	{
		memcpy((void*)(Hst_tempPeds[GetCurrentStream()]+Data_block_size*scenarioID),
				(const void*)hst_particle->peds,
				Data_block_size*sizeof(Dvc_PedStruct));
	}
	else
	{
		memcpy((void*)(Hst_tempPeds[0]+Data_block_size*scenarioID),
				(const void*)hst_particle->peds,
				Data_block_size*sizeof(Dvc_PedStruct));
	}
}
HOST void Dvc_PomdpState::CopyPedsToGPU(Dvc_PomdpState* dvc_particles, int NumParticles, bool deep_copy)
{
	if(deep_copy)
	{
		int Data_size=NumParticles*ModelParams::N_PED_IN;
		dim3 grid1(NumParticles,1);dim3 threads1(ModelParams::N_PED_IN,1);
		if(Globals::config.use_multi_thread_ && StreamManager::MANAGER.cuda_streams)
		{
			HANDLE_ERROR(hipMemcpyAsync((void*)Dvc_tempPeds[GetCurrentStream()],
					(const void*)Hst_tempPeds[GetCurrentStream()],
					Data_size*sizeof(Dvc_PedStruct),
					hipMemcpyHostToDevice,((hipStream_t*)StreamManager::MANAGER.cuda_streams)[GetCurrentStream()]));
			logd << "dvc_particles=" << dvc_particles<< ",Dvc_tempPeds[i]=" << Dvc_tempPeds[GetCurrentStream()] <<", GetCurrentStream()="<< GetCurrentStream()<< endl;

			CopyPeds_to_Particles<<<grid1, threads1, 0, ((hipStream_t*)StreamManager::MANAGER.cuda_streams)[GetCurrentStream()]>>>
					(dvc_particles,Dvc_tempPeds[GetCurrentStream()]);
		}
		else
		{
			HANDLE_ERROR(hipMemcpy((void*)Dvc_tempPeds[0],
					(const void*)Hst_tempPeds[0],
					Data_size*sizeof(Dvc_PedStruct),
					hipMemcpyHostToDevice));

			logd << "dvc_particles=" << dvc_particles<< ",Dvc_tempPeds[0]=" << Dvc_tempPeds[0]<< endl;
			CopyPeds_to_Particles<<<grid1, threads1>>>(dvc_particles,Dvc_tempPeds[0]);
		}

		//HANDLE_ERROR( hipDeviceSynchronize());

	}
}

HOST void Dvc_PomdpState::ReadMainStateBackToCPU(const Dvc_PomdpState* dvc_particles, PomdpState* hst_particle)
{
	int ThreadID=0;
	if(Globals::config.use_multi_thread_)
		ThreadID=Globals::MapThread(this_thread::get_id());
	HANDLE_ERROR(hipMemcpy((void*)Hst_temp_mainstates[ThreadID], (const void*)dvc_particles, sizeof(Dvc_PomdpState), hipMemcpyDeviceToHost));
	hst_particle->car.dist_travelled=Hst_temp_mainstates[ThreadID]->car.dist_travelled;
	hst_particle->car.pos=Hst_temp_mainstates[ThreadID]->car.pos;
	hst_particle->car.vel=Hst_temp_mainstates[ThreadID]->car.vel;

	hst_particle->num=Hst_temp_mainstates[ThreadID]->num;
	hst_particle->weight=Hst_temp_mainstates[ThreadID]->weight;
	hst_particle->state_id=Hst_temp_mainstates[ThreadID]->state_id;
	hst_particle->scenario_id=Hst_temp_mainstates[ThreadID]->scenario_id;
}

HOST void Dvc_PomdpState::ReadPedsBackToCPU(const Dvc_PomdpState* dvc_particles,
		std::vector<State*> hst_particles, bool deep_copy)
{
	if(deep_copy)
	{
		int ThreadID=0;
		if(Globals::config.use_multi_thread_)
			ThreadID=Globals::MapThread(this_thread::get_id());

		int NumParticles=hst_particles.size();
		int Data_size=NumParticles*ModelParams::N_PED_IN;
		dim3 grid1(NumParticles,1);dim3 threads1(ModelParams::N_PED_IN,1);
		if(Globals::config.use_multi_thread_ && StreamManager::MANAGER.cuda_streams)
		{
			CopyPeds_to_list<<<grid1, threads1, 0, ((hipStream_t*)StreamManager::MANAGER.cuda_streams)[ThreadID]>>>
					(dvc_particles,Dvc_tempPeds[ThreadID]);
			HANDLE_ERROR(hipMemcpyAsync((void*)Hst_tempPeds[ThreadID],
					(const void*)Dvc_tempPeds[ThreadID],
					Data_size*sizeof(Dvc_PedStruct),
					hipMemcpyDeviceToHost,((hipStream_t*)StreamManager::MANAGER.cuda_streams)[ThreadID]));
			hipStreamSynchronize(((hipStream_t*)StreamManager::MANAGER.cuda_streams)[ThreadID]);
		}
		else
		{
			CopyPeds_to_list<<<grid1, threads1>>>(dvc_particles,Dvc_tempPeds[0]);
			HANDLE_ERROR(hipMemcpy((void*)Hst_tempPeds[0],
					(const void*)Dvc_tempPeds[0],
					Data_size*sizeof(Dvc_PedStruct),
					hipMemcpyDeviceToHost));
		}


		int Data_block_size=ModelParams::N_PED_IN;

		for(int i=0;i<NumParticles;i++)
		{
			PomdpState* car_state=static_cast<PomdpState*>(hst_particles[i]);

			if(Globals::config.use_multi_thread_ && StreamManager::MANAGER.cuda_streams)
			{
				memcpy((void*)car_state->peds,
						(const void*)(Hst_tempPeds[ThreadID]+Data_block_size*i),
						Data_block_size*sizeof(Dvc_PedStruct));
			}
			else
			{
				memcpy((void*)car_state->peds,
						(const void*)(Hst_tempPeds[0]+Data_block_size*i),
						Data_block_size*sizeof(Dvc_PedStruct));
			}
		}
	}
}


__global__ void CopyParticles(Dvc_PomdpState* des,Dvc_PomdpState* src,
		float* weight,int* particle_IDs,int num_particles,
		Dvc_RandomStreams* streams, int stream_pos
		)
{
	int pos=blockIdx.x*blockDim.x+threadIdx.x;

	if(pos==0)
	{
		weight[0]=0;
		if(streams) streams->position_=stream_pos;
	}
	if(pos < num_particles)
	{

		int scenarioID=particle_IDs[pos];
		Dvc_PomdpState* src_i=src+scenarioID;//src is a full length array for all particles
		Dvc_PomdpState* des_i=des+pos;//des is short, only for the new partition

		des_i->car.dist_travelled=src_i->car.dist_travelled;
		des_i->car.pos=src_i->car.pos;
		des_i->car.vel=src_i->car.vel;
		des_i->num=src_i->num;
		des_i->weight=src_i->weight;
		des_i->state_id=src_i->state_id;
		des_i->scenario_id=src_i->scenario_id;

		for(int i=0;i<src_i->num;i++)
		{
			des_i->peds[i].goal=src_i->peds[i].goal;
			des_i->peds[i].id=src_i->peds[i].id;
			des_i->peds[i].pos.x=src_i->peds[i].pos.x;
			des_i->peds[i].pos.y=src_i->peds[i].pos.y;
			des_i->peds[i].vel=src_i->peds[i].vel;
		}

		if(!Dvc_config->track_alpha_vector)
		{
			//Accumulate weight of the particles
			atomicAdd(weight, des_i->weight);
		}
	}
}

void PedPomdp::CreateMemoryPool() const
{
	if(gpu_mainstate_pool_==NULL)
		gpu_mainstate_pool_=new GPU_MemoryPool<Dvc_PomdpState>;
	if(gpu_ped_pool_==NULL)
		gpu_ped_pool_=new GPU_MemoryPool<Dvc_PedStruct>;
}

void PedPomdp::DestroyMemoryPool(MEMORY_MODE mode) const
{
	switch(mode)
	{
		case DESTROY:
			if(gpu_mainstate_pool_){delete gpu_mainstate_pool_;gpu_mainstate_pool_=NULL;}
			if(gpu_ped_pool_){delete gpu_ped_pool_;gpu_ped_pool_=NULL;}
			break;
		case RESET:
			if(gpu_mainstate_pool_ ){ gpu_mainstate_pool_->ResetChuncks();};
			if(gpu_ped_pool_ ){ gpu_ped_pool_->ResetChuncks();};
			break;
	}
}
__global__ void LinkPeds(Dvc_PomdpState* state, Dvc_PedStruct* peds_memory, int numParticles)
{
	for(int i=0;i<numParticles;i++)
	{
		state[i].peds=peds_memory+i*Dvc_ModelParams::N_PED_IN;
	}
}

Dvc_State* PedPomdp::AllocGPUParticles(int numParticles, MEMORY_MODE mode, Dvc_State*** particles_for_all_actions) const
{
	clock_t start=clock();
	dim3 grid((numParticles+THREADDIM-1)/THREADDIM,1); dim3 threads(THREADDIM,1);
	int num_threads=1;

	if(Globals::config.use_multi_thread_)
	{
		num_threads = Globals::config.NUM_THREADS;
	}

	Dvc_PedStruct* node_particle_peds;
	switch(mode)
	{
	case INIT:

		CreateMemoryPool();

		/* Intermediate pedestrian container for copying pedestrians in host particles to device particles */
		if(Dvc_tempPeds == NULL && Hst_tempPeds == NULL){
			Dvc_tempPeds=new Dvc_PedStruct*[num_threads];
			Hst_tempPeds=new Dvc_PedStruct*[num_threads];
			for(int i=0;i<num_threads;i++)
			{
				HANDLE_ERROR(hipMalloc((void**)&Dvc_tempPeds[i],numParticles*ModelParams::N_PED_IN*sizeof(Dvc_PedStruct) ));
				HANDLE_ERROR(hipHostAlloc((void**)&Hst_tempPeds[i],numParticles*ModelParams::N_PED_IN*sizeof(Dvc_PedStruct),0 ));
			}
		}

		cout<<"numParticles="<<numParticles<<endl;

		if(particles_for_all_actions[0] == NULL){
			particles_for_all_actions[0]=new Dvc_State*[num_threads];
			//Allocate pedestrian memory separately
			Dvc_PedStruct*  peds_tmp=gpu_ped_pool_->Allocate((NumActions()*num_threads)*numParticles*ModelParams::N_PED_IN);

			for(int i=0;i<num_threads;i++)
			{
				HANDLE_ERROR(hipMalloc((void**)&particles_for_all_actions[0][i],
						NumActions()*numParticles*sizeof(Dvc_PomdpState)));
				//Link pre-allocated pedestrian memory 
				LinkPeds<<<dim3(numParticles,1), dim3(ModelParams::N_PED_IN,1)>>>
						(static_cast<Dvc_PomdpState*>(particles_for_all_actions[0][i]),
						peds_tmp+(NumActions()*i)*numParticles*ModelParams::N_PED_IN,
						NumActions()*numParticles);
			}
			//Record the ped memory used by the pre-allocated lists
			//never reuse these memory for vnode particles
			gpu_ped_pool_->RecordHead();
		}

		/*Intermediate memory for copying particle IDs to device memory 
		hipHostAlloc enables the copying to interleave with kernel executions*/
		Hst_temp_IDs=new int*[num_threads];
		for(int i=0;i<num_threads;i++)
		{
			if(Globals::config.track_alpha_vector)
			{
				hipHostAlloc(&Hst_temp_IDs[i],(2+ Globals::config.num_scenarios + Globals::config.num_obs)*NumActions()*sizeof(int),0);
			}
			else
			{
				hipHostAlloc(&Hst_temp_IDs[i],numParticles*sizeof(int),0);
			}
		}

		/*Intermediate memory for copying weights to device memory. 
		hipHostAlloc enables the copying to interleave with kernel executions*/

		Hst_temp_weight=new float*[num_threads];
		for(int i=0;i<num_threads;i++)
			hipHostAlloc(&Hst_temp_weight[i],1*sizeof(float),0);

		Dvc_temp_weight=new float*[num_threads];
		for(int i=0;i<num_threads;i++)
			HANDLE_ERROR(hipMalloc(&Dvc_temp_weight[i], sizeof(float)));


		/*Intermediate memory for copying main memory of particle (everything except pedestrians) from device back to host
		hipHostAlloc enables the copying to interleave with kernel executions*/
		Hst_temp_mainstates=new Dvc_PomdpState*[num_threads];

		for(int i=0;i<num_threads;i++)
			HANDLE_ERROR(hipHostAlloc((void**)&Hst_temp_mainstates[i],1*sizeof(Dvc_PomdpState),0));

		/* No node particle allocated */
		return NULL;

	case ALLOC_ROOT:

		/*Intermediate managed memory for root node particles.
		 * Managed memory enables data copying between CPU and GPU without launching memcpy (which is expensive)
		 */
		HANDLE_ERROR(hipMallocManaged((void**)&Managed_rootnode_particles, numParticles*sizeof(Dvc_PomdpState)));

		node_particle_peds = gpu_ped_pool_->Allocate(numParticles*ModelParams::N_PED_IN);

		/* Link pedestrian lists to the main memory of particles */
		LinkPeds<<<dim3(numParticles,1), dim3(ModelParams::N_PED_IN,1)>>>(Managed_rootnode_particles, node_particle_peds, numParticles);
		HANDLE_ERROR(hipDeviceSynchronize());
		return Managed_rootnode_particles;

	case ALLOC:

		/* Allocate vnode particles: main memory and the pedestrian lists */
		Dvc_PomdpState* vnode_particles = gpu_mainstate_pool_->Allocate(numParticles);
		Dvc_PedStruct* vnode_particle_peds = gpu_ped_pool_->Allocate(numParticles*ModelParams::N_PED_IN);

		/* Link pedestrian lists to the main memory of particles */
		LinkPeds<<<dim3(numParticles,1), dim3(ModelParams::N_PED_IN,1)>>>(vnode_particles, vnode_particle_peds, numParticles);
		HANDLE_ERROR(hipDeviceSynchronize());
		return vnode_particles;
	};


	return NULL;
}


void PedPomdp::CopyGPUParticlesFromParent(Dvc_State* des,Dvc_State* src,int src_offset,
		int* dvc_particle_IDs,int num_particles,bool interleave,
		Dvc_RandomStreams* streams, int stream_pos,
		void* cudaStream, int shift) const
{
	dim3 grid((num_particles+THREADDIM-1)/THREADDIM,1); dim3 threads(THREADDIM,1);
	if(num_particles<THREADDIM)
	{
		grid.x=1;grid.y=1;threads.x=num_particles;
	}

	int ThreadID=0;
	if(Globals::config.use_multi_thread_)
		ThreadID=Globals::MapThread(this_thread::get_id());
	if(cudaStream)
	{
		CopyParticles<<<grid, threads,0, *(hipStream_t*)cudaStream>>>(static_cast<Dvc_PomdpState*>(des),
				static_cast<Dvc_PomdpState*>(src)+src_offset,Dvc_temp_weight[(ThreadID+shift)%Globals::config.NUM_THREADS],
				dvc_particle_IDs,num_particles, streams,stream_pos);
		if(!interleave)
			;
	}
	else
	{
		CopyParticles<<<grid, threads,0, 0>>>(static_cast<Dvc_PomdpState*>(des),
				static_cast<Dvc_PomdpState*>(src)+src_offset,Dvc_temp_weight[ThreadID],
				dvc_particle_IDs,num_particles, streams,stream_pos);
		if(!interleave)
			HANDLE_ERROR(hipDeviceSynchronize());
	}
}


Dvc_State* PedPomdp::GetPointerToParticleList(int offset,  Dvc_State* full_list) const
{
	return static_cast<Dvc_PomdpState*>(full_list)+ offset;
}
Dvc_State* PedPomdp::CopyParticlesToGPU(Dvc_State* dvc_particles, const std::vector<State*>& particles, bool deep_copy) const
	//dvc_particles: managed device memory storing particles
	// deep_copy: option on whether to copy list objects in side particles
{

	auto start = Time::now();


	for (int i=0;i<particles.size();i++)
	{
		const PomdpState* src=static_cast<const PomdpState*>(particles[i]);
		Dvc_PomdpState::CopyMainStateToGPU(static_cast<const Dvc_PomdpState*>(dvc_particles),src->scenario_id,src);
	}
	Dvc_PomdpState::CopyPedsToGPU(static_cast<const Dvc_PomdpState*>(dvc_particles),particles.size());

	return dvc_particles;
}

void PedPomdp::CopyParticleIDsToGPU( int* Dvc_ptr, const std::vector<int>& particleIDs, void *cudaStream) const
{
	if(cudaStream)
	{
		int ThreadID=Globals::MapThread(this_thread::get_id());
		memcpy(Hst_temp_IDs[ThreadID],particleIDs.data(),particleIDs.size()*sizeof(int));

		HANDLE_ERROR(hipMemcpyAsync(Dvc_ptr,Hst_temp_IDs[ThreadID],particleIDs.size()*sizeof(int), hipMemcpyHostToDevice,*(hipStream_t*)cudaStream));
	}
	else
	{
		logd << "Dvc_ptr = "<< Dvc_ptr << " particleIDs.size() = " << particleIDs.size()<< " cudaStream = "<< cudaStream<< endl;
		HANDLE_ERROR(hipMemcpy(Dvc_ptr,particleIDs.data(),particleIDs.size()*sizeof(int), hipMemcpyHostToDevice));
	}
}


void PedPomdp::DeleteGPUParticles( MEMORY_MODE mode, Dvc_State** particles_for_all_actions ) const
{
	int num_threads=1;

	switch (mode){
	case DESTROY:

		if(Globals::config.use_multi_thread_)
		{
			num_threads=Globals::config.NUM_THREADS;
		}
		for(int i=0;i<num_threads;i++)
		{
			if(particles_for_all_actions[i]!=NULL)
				{HANDLE_ERROR(hipFree(particles_for_all_actions[i]));particles_for_all_actions[i]=NULL;}
		}
		if(particles_for_all_actions)delete [] particles_for_all_actions;particles_for_all_actions=NULL;
		for(int i=0;i<num_threads;i++)
		{
			hipHostFree(Hst_temp_IDs[i]);
		}
		delete [] Hst_temp_IDs;
		for(int i=0;i<num_threads;i++)
		{
			hipHostFree(Hst_temp_weight[i]);
		}
		delete [] Hst_temp_weight;
		for(int i=0;i<num_threads;i++)
		{
			hipFree(Dvc_temp_weight[i]);
		}
		delete [] Dvc_temp_weight;

		for(int i=0;i<num_threads;i++)
		{
			hipFree(Dvc_tempPeds[i]);
			hipHostFree(Hst_tempPeds[i]);
			hipHostFree(Hst_temp_mainstates[i]);
		}

		delete [] Dvc_tempPeds;
		delete [] Hst_tempPeds;
		delete [] Hst_temp_mainstates;
		break;
	case RESET:
		HANDLE_ERROR(hipFree(static_cast<Dvc_PomdpState*>(Managed_rootnode_particles)));

		break;
	};

	DestroyMemoryPool(mode);
}


DEVICE float Dvc_PedPomdpParticleUpperBound1::Value(
		const Dvc_State* particles, int scenarioID, Dvc_History& history) {

	return Dvc_ModelParams::GOAL_REWARD / (1 - Dvc_Globals::Dvc_Discount(Dvc_config));
}



DEVICE bool Dvc_PedPomdp::Dvc_Step(Dvc_State& state, float rand_num, int action, float& reward,
	int* obs) {

	Dvc_PomdpState& pedpomdp_state = static_cast<Dvc_PomdpState&>(state);//copy contents, link cells to existing ones
	__shared__ int iscollision[32];

	if(FIX_SCENARIO==1 || GPUDoPrint)
		if(GPUDoPrint && pedpomdp_state.scenario_id==PRINT_ID && blockIdx.x==ACTION_ID && threadIdx.y==0){
			printf("(GPU) Before step: scenario=%d \n", pedpomdp_state.scenario_id);
			printf("action= %d\n ",action);
			printf("Before step:\n");
			int pos=pedpomdp_state.car.pos;
			printf("car_pox= %d ",pos);
			printf("trav_dist=%f\n",pedpomdp_state.car.dist_travelled);
			printf("car_vel= %f\n",pedpomdp_state.car.vel);

			for(int i=0;i<pedpomdp_state.num;i++)
			{
				printf("ped %d pox_x= %f pos_y=%f\n",i,
						pedpomdp_state.peds[i].pos.x,pedpomdp_state.peds[i].pos.y);
			}
		}

	bool terminal=false;
	reward = 0;

	unsigned long long int Temp=INIT_QUICKRANDSEED;

	/* Termination checking */
	if(threadIdx.y==0)
	{
		// Terminate upon reaching goal
		if (pedpomdp_state.car.dist_travelled > Dvc_ModelParams::GOAL_TRAVELLED-1e-4
				|| pedpomdp_state.car.pos >= path->size_-1) {
			reward = Dvc_ModelParams::GOAL_REWARD;
			terminal= true;
		}
	}

	/* Collision checking */
	iscollision[threadIdx.x]=false;
	__syncthreads();

	if(!terminal)
	{
		const int car = pedpomdp_state.car.pos;
		const Dvc_COORD& car_pos = path->way_points_[car];
		const Dvc_COORD& forward_pos = path->way_points_[path->forward(car, 1.0)];

		if(threadIdx.y<pedpomdp_state.num){
			const Dvc_COORD& pedpos = pedpomdp_state.peds[threadIdx.y].pos;
			bool collide_ped=false;
			float HNx = forward_pos.x - car_pos.x, // car direction
						 HNy = forward_pos.y - car_pos.y;
			float HMx = pedpos.x - car_pos.x,
						 HMy = pedpos.y - car_pos.y;


/// car geomery for golfcart
/*			double car_width = 0.87,
			car_length = 1.544;

			double safe_margin = 0.92, side_safe_margin = 0.4, back_safe_margin = 0.33,
				 side_margin = car_width / 2.0 + side_safe_margin,
				 front_margin = car_length/2.0 + safe_margin,
				 back_margin = car_length/2.0 + back_safe_margin;
*/
/// end golfcart

/// car geomery for audi r8
			/*double car_width = 2.0,
			 car_length = 4.4;

			 double safe_margin = 0.8, side_safe_margin = 0.35, back_safe_margin = 0.2,
			 side_margin = car_width / 2.0 + side_safe_margin,
			 front_margin = 3.6 + safe_margin,
			 back_margin = 0.8 + back_safe_margin;*/
/// end audi r8


/// car geometry for pomdp car
			double car_width = 1.2,
						 car_length = 2.2;
			double safe_margin = 0.3,
				 side_margin = car_width / 2.0 + safe_margin,
				 front_margin = safe_margin,
				 back_margin = car_length + safe_margin;
/// end pomdp car


			float HLx = - HNy, // direction after 90 degree anticlockwise rotation
						 HLy = HNx;

			float HM_HN = HMx * HNx + HMy * HNy, // HM . HN
						 HN_HN = HNx * HNx + HNy * HNy; // HN . HN
			if (HM_HN >= 0 && HM_HN * HM_HN > HN_HN * front_margin * front_margin)
				collide_ped = false;
			else if (HM_HN <= 0 && HM_HN * HM_HN > HN_HN * back_margin * back_margin)
				collide_ped = false;
			else
			{
			    float HM_HL = HMx * HLx + HMy * HLy, // HM . HL
						 HL_HL = HLx * HLx + HLy * HLy; // HL . HL
			    collide_ped= HM_HL * HM_HL <= HL_HL * side_margin * side_margin;
			}
			atomicOr(iscollision+threadIdx.x, collide_ped);
		}
	}
	__syncthreads(); // Synchronize the block to wait for collision checking with all peds (parallelized in the Y dimemsion) to finish.

	if(threadIdx.y==0 && !terminal)
	{

		/* Terminate if collision is detected */
		if(pedpomdp_state.car.vel > 0.001 && iscollision[threadIdx.x] ) { /// collision occurs only when car is moving
		    reward= Dvc_ModelParams::CRASH_PENALTY *
		    		(pedpomdp_state.car.vel * pedpomdp_state.car.vel +
		    				Dvc_ModelParams::REWARD_BASE_CRASH_VEL);

		    if(action == ACT_DEC) reward += 0.1;

			terminal= true;
		}

		/* Compute reward */
		if(!terminal)
		{
			// Smoothness penalty
			reward += (action == ACT_DEC || action == ACT_ACC) ? -0.1 : 0.0;

			reward += Dvc_ModelParams::REWARD_FACTOR_VEL *
					(pedpomdp_state.car.vel - Dvc_ModelParams::VEL_MAX) / Dvc_ModelParams::VEL_MAX;

			float acc = (action == ACT_ACC) ? Dvc_ModelParams::AccSpeed :
				((action == ACT_CUR) ?  0 : (-Dvc_ModelParams::AccSpeed));

			/* State transition: car */
			float dist = pedpomdp_state.car.vel / freq;
			int nxt = path->forward(pedpomdp_state.car.pos, dist);
			pedpomdp_state.car.pos = nxt;
			pedpomdp_state.car.dist_travelled += dist;

			const float N = Dvc_ModelParams::NOISE_ROBVEL;
			if (N>0) {
				if(FIX_SCENARIO!=1 && !GPUDoPrint)
					rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);

				float prob = rand_num;
				if (prob > N) {
					pedpomdp_state.car.vel += acc / freq;
				}
			} else {
				pedpomdp_state.car.vel += acc / freq;
			}
			pedpomdp_state.car.vel = max(min(pedpomdp_state.car.vel, Dvc_ModelParams::VEL_MAX), 0.0);
		}
	}
	__syncthreads();


	if(!terminal)
	{
		/* State transition: peds */
		if(threadIdx.y<pedpomdp_state.num)
		{
			int i=0;
			while(i<threadIdx.y)
			{
				if(FIX_SCENARIO!=1 && !GPUDoPrint)
					rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
				i++;
			}
			if(threadIdx.y!=0 && FIX_SCENARIO!=1 && !GPUDoPrint)
				rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);

			const Dvc_COORD& goal = goals[pedpomdp_state.peds[threadIdx.y].goal];
			if (abs(goal.x+1)<1e-5 && abs(goal.y+1)<1e-5) {  //stop intention, ped doesn't move
				;
			}
			else
			{
				// Straightline model with Gussian noise on directions
				Dvc_Vector goal_vec(goal.x - pedpomdp_state.peds[threadIdx.y].pos.x, goal.y - pedpomdp_state.peds[threadIdx.y].pos.y);
				float a = goal_vec.GetAngle();
				float noise = sqrt(-2 * log(rand_num));

				if(FIX_SCENARIO!=1 && !GPUDoPrint)
					rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
				noise *= cos(2 * M_PI * rand_num)* Dvc_ModelParams::NOISE_GOAL_ANGLE;
				a += noise;

				Dvc_Vector move(a, pedpomdp_state.peds[threadIdx.y].vel/freq, 0);
				pedpomdp_state.peds[threadIdx.y].pos.x += move.dw;
				pedpomdp_state.peds[threadIdx.y].pos.y += move.dh;
			}
		}
	}
	__syncthreads();


	if(threadIdx.y==0 && obs!=NULL)//for each particle in the thread block
	{
		/* generate observations by descretizing the observable part of the state */
		if(!terminal)
		{
			int i=0;
			obs[i++]=2+2*pedpomdp_state.num;
			obs[i++] = int(pedpomdp_state.car.pos);
			obs[i++] = int((pedpomdp_state.car.vel+1e-5) / Dvc_ModelParams::vel_rln);
			for(int j = 0; j < pedpomdp_state.num; j ++) {
				obs[i++] = int(pedpomdp_state.peds[j].pos.x / Dvc_ModelParams::pos_rln);
				obs[i++] = int(pedpomdp_state.peds[j].pos.y / Dvc_ModelParams::pos_rln);
			}
		}
		else
		{
			int i=0;
			obs[i++]=0;
			obs[i++] = 0;
			obs[i++] = 0;
			for(int j = 0; j < pedpomdp_state.num; j ++) {
				obs[i++] = 0;
				obs[i++] = 0;
			}
		}
	}

	if(!terminal && GPUDoPrint && pedpomdp_state.scenario_id==PRINT_ID && blockIdx.x==ACTION_ID && threadIdx.y==0){
		printf("(GPU) After step: scenario=%d \n", pedpomdp_state.scenario_id);
		printf("rand=%f, action=%d \n", rand_num, action);
		printf("After step:\n");
		printf("Reward=%f\n",reward);
		int pos=pedpomdp_state.car.pos;
		printf("car pox= %d ",pos);
		printf("dist=%f\n",pedpomdp_state.car.dist_travelled);
		printf("car vel= %f\n",pedpomdp_state.car.vel);
		for(int i=0;i<pedpomdp_state.num;i++)
		{
			printf("ped %d pox_x= %f pos_y=%f\n",i,
					pedpomdp_state.peds[i].pos.x,pedpomdp_state.peds[i].pos.y);
		}
	}
	return terminal;
}

DEVICE int Dvc_PedPomdp::NumActions() {
	return 3;
}


DEVICE float Dvc_PedPomdp::Dvc_ObsProbInt(int* obs, Dvc_State& state, int action)
{
	//const PomdpState& state = static_cast<const PomdpState&>(s);
	Dvc_PomdpState& pedpomdp_state = static_cast<Dvc_PomdpState&>(state);//copy contents, link cells to existing ones
		//PrintState(state);
		float prob = 1.0;
		float b = 0.0;
		for (int j = 0; j < pedpomdp_state.num; j ++) {
		  b = b + ((obs[2*j + 3]*Dvc_ModelParams::pos_rln) - pedpomdp_state.peds[j].pos.x )*((obs[2*j + 3]*Dvc_ModelParams::pos_rln) - pedpomdp_state.peds[j].pos.x );
		  b = b + ((obs[2*j + 4]*Dvc_ModelParams::pos_rln) - pedpomdp_state.peds[j].pos.y )*((obs[2*j + 4]*Dvc_ModelParams::pos_rln) - pedpomdp_state.peds[j].pos.y );
		  //std::cout << j << " obs vec " << obs[2*j + 2]<< "," << obs[2*j + 3] << ")b= " << b<< std::endl;
		}
		float stddev = 1.0;
		b = - b / (2.0* stddev*stddev);
		//std::cout << "b= " << b << std::endl;
		return expf(b);
}
DEVICE void Dvc_PedPomdp::Dvc_Copy_NoAlloc(Dvc_State* des, const Dvc_State* src, int pos, bool offset_des) {
	/*Pass member values, assign member pointers to existing state pointer*/
	const Dvc_PomdpState* src_i= static_cast<const Dvc_PomdpState*>(src)+pos;
	if(!offset_des) pos=0;
	Dvc_PomdpState* des_i= static_cast<const Dvc_PomdpState*>(des)+pos;
	des_i->weight=src_i->weight;
	des_i->scenario_id=src_i->scenario_id;
	des_i->num=src_i->num;
	des_i->car.dist_travelled=src_i->car.dist_travelled;
	des_i->car.pos=src_i->car.pos;
	des_i->car.vel=src_i->car.vel;
	for(int i=0;i< des_i->num;i++)
	{
		des_i->peds[i].vel=src_i->peds[i].vel;
		des_i->peds[i].pos.x=src_i->peds[i].pos.x;
		des_i->peds[i].pos.y=src_i->peds[i].pos.y;
		des_i->peds[i].goal=src_i->peds[i].goal;
		des_i->peds[i].id=src_i->peds[i].id;
	}
}

DEVICE void Dvc_PedPomdp::Dvc_Copy_ToShared(Dvc_State* des, const Dvc_State* src, int pos, bool offset_des) {
	/*Pass member values, assign member pointers to existing state pointer*/
	const Dvc_PomdpState* src_i= static_cast<const Dvc_PomdpState*>(src)+pos;
	if(!offset_des) pos=0;
	Dvc_PomdpState* des_i= static_cast<const Dvc_PomdpState*>(des)+pos;
	des_i->weight=src_i->weight;
	des_i->scenario_id=src_i->scenario_id;
	des_i->num=src_i->num;
	des_i->car.dist_travelled=src_i->car.dist_travelled;
	des_i->car.pos=src_i->car.pos;
	des_i->car.vel=src_i->car.vel;
	des_i->peds=(Dvc_PedStruct*)((void*)(des_i)+3*sizeof(Dvc_PedStruct));
	for(int i=0;i< des_i->num;i++)
	{
		des_i->peds[i].vel=src_i->peds[i].vel;
		des_i->peds[i].pos.x=src_i->peds[i].pos.x;
		des_i->peds[i].pos.y=src_i->peds[i].pos.y;
		des_i->peds[i].goal=src_i->peds[i].goal;
		des_i->peds[i].id=src_i->peds[i].id;
	}
}
DEVICE Dvc_State* Dvc_PedPomdp::Dvc_Get(Dvc_State* particles, int pos) {
	Dvc_PomdpState* particle_i= static_cast<Dvc_PomdpState*>(particles)+pos;

	return particle_i;
}

DEVICE Dvc_ValuedAction Dvc_PedPomdp::Dvc_GetBestAction() {
	return Dvc_ValuedAction(0,
			Dvc_ModelParams::CRASH_PENALTY * (Dvc_ModelParams::VEL_MAX*Dvc_ModelParams::VEL_MAX + Dvc_ModelParams::REWARD_BASE_CRASH_VEL));
}

void PedPomdp::ReadParticlesBackToCPU(std::vector<State*>& particles ,const Dvc_State* dvc_particles,
			bool deepcopy) const
{
	for (int i=0;i<particles.size();i++)
	{
		const Dvc_PomdpState* src=static_cast<const Dvc_PomdpState*>(dvc_particles)+i;
		PomdpState* des=static_cast<PomdpState*>(particles[i]);
		Dvc_PomdpState::ReadMainStateBackToCPU(src,des);
	}
	Dvc_PomdpState::ReadPedsBackToCPU(
			static_cast<const Dvc_PomdpState*>(dvc_particles),
			particles);
}
