#include "hip/hip_runtime.h"
#include "GPU_UncNavigation.h"

#include "GPU_base_unc_navigation.h"
#include <base_unc_navigation.h>
#include <despot/GPUutil/GPUrandom.h>


using namespace std;

namespace despot {

DEVICE int num_obs_bits = 8;
DEVICE float OBS_NOISE = 0.03f;
DEVICE Dvc_UncNavigation::Dvc_UncNavigation()

{

}


DEVICE Dvc_UncNavigation::~Dvc_UncNavigation()
{
}

DEVICE bool Dvc_UncNavigation::Dvc_Step(Dvc_State& state, float rand_num, int action, float& reward,
	OBS_TYPE& obs) {

	Dvc_UncNavigationState& nav_state = static_cast<Dvc_UncNavigationState&>(state);//copy contents, link cells to existing ones
	bool terminal=false;
	reward = 0;

	int dir=threadIdx.y;

	if(dir==0)
	{
		terminal=(nav_state.rob==nav_state.goal);

		reward=-0.1;// small cost for one step
		DvcCoord rob_pos=nav_state.rob;

		float prob=1.0f-STEP_NOISE;

		if (action < E_STAY && terminal!=true) { // Move
			// only succeed with 80% chance
			rob_pos +=(rand_num<prob)? Dvc_Compass::GetDirections(action):DvcCoord(0,0);
			bool validmove=(nav_state.Inside(rob_pos) && nav_state.CollisionCheck(rob_pos)==false);

			nav_state.rob=validmove?rob_pos:nav_state.rob;
			reward=validmove?-0.1:-1;
			reward=(nav_state.rob==nav_state.goal)?/*10*/GOAL_REWARD:reward;
		}

		if (action == E_STAY) { // Sample
			reward=-0.2;
		}

		obs=0;//Initialize obs
	}

	OBS_TYPE obs_i=0;

	unsigned long long int Temp=INIT_QUICKRANDSEED;
	for(dir=0;dir<num_obs_bits;dir++)
	{
		switch(dir)
		{
		case 3:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x,nav_state.rob.y+1):!nav_state.Grid(nav_state.rob.x,nav_state.rob.y+1);
			break;
		case 2:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x+1,nav_state.rob.y):!nav_state.Grid(nav_state.rob.x+1,nav_state.rob.y);
			break;
		case 1:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x,nav_state.rob.y-1):!nav_state.Grid(nav_state.rob.x,nav_state.rob.y-1);
			break;
		case 0:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x-1,nav_state.rob.y):!nav_state.Grid(nav_state.rob.x-1,nav_state.rob.y);
			break;
		case 4:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x-1,nav_state.rob.y+1):!nav_state.Grid(nav_state.rob.x-1,nav_state.rob.y+1);
			break;
		case 5:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x-1,nav_state.rob.y-1):!nav_state.Grid(nav_state.rob.x-1,nav_state.rob.y-1);
			break;
		case 6:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x+1,nav_state.rob.y-1):!nav_state.Grid(nav_state.rob.x+1,nav_state.rob.y-1);
			break;
		case 7:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x+1,nav_state.rob.y+1):!nav_state.Grid(nav_state.rob.x+1,nav_state.rob.y+1);
			break;
		case 11:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x,nav_state.rob.y+2):!nav_state.Grid(nav_state.rob.x,nav_state.rob.y+2);
			break;
		case 10:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x+2,nav_state.rob.y):!nav_state.Grid(nav_state.rob.x+2,nav_state.rob.y);
			break;
		case 9:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x,nav_state.rob.y-2):!nav_state.Grid(nav_state.rob.x,nav_state.rob.y-2);
			break;
		case 8:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x-2,nav_state.rob.y):!nav_state.Grid(nav_state.rob.x-2,nav_state.rob.y);
			break;
		case 12:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x-2,nav_state.rob.y+2):!nav_state.Grid(nav_state.rob.x-2,nav_state.rob.y+2);
			break;
		case 13:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x-2,nav_state.rob.y-2):!nav_state.Grid(nav_state.rob.x-2,nav_state.rob.y-2);
			break;
		case 14:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x+2,nav_state.rob.y-2):!nav_state.Grid(nav_state.rob.x+2,nav_state.rob.y-2);
			break;
		case 15:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x+2,nav_state.rob.y+2):!nav_state.Grid(nav_state.rob.x+2,nav_state.rob.y+2);
			break;
		}
		obs=(obs|(obs_i<<dir));
	}

	if(obs>=Dvc_NumObservations())
		printf("Wrong obs %d", obs);

	if(threadIdx.y==0)
	{
		if(terminal){reward=0;obs=Dvc_NumObservations()-1;}
	}
	return terminal;
}

DEVICE float Dvc_UncNavigation::Dvc_ObsProb(OBS_TYPE& obs, Dvc_State& state, int action)
{
		float prob=1;
		Dvc_UncNavigationState nav_state(static_cast<const Dvc_UncNavigationState&>(state));

		OBS_TYPE my_obs = obs;
		int obs_North=(my_obs%16)/8;
		int obs_East=((my_obs%16)-obs_North*8)/4;
		int obs_South=((my_obs%16)-obs_North*8-obs_East*4)/2;
		int obs_West=((my_obs%16)-obs_North*8-obs_East*4-obs_South*2);
		my_obs = my_obs/16;
		int obs_North_East= (my_obs%16)/8;
		int obs_South_East=((my_obs % 16)-obs_North_East*8)/4;
		int obs_South_West=((my_obs % 16)-obs_North_East*8-obs_South_East*4)/2;
		int obs_North_West=((my_obs % 16)-obs_North_East*8-obs_South_East*4-obs_South_West*2);
		int obs_North2, obs_East2, obs_South2, obs_West2, obs_North_East2, obs_South_East2, obs_South_West2, obs_North_West2;

		if(num_obs_bits > 8)
		{
			my_obs = my_obs/16;
			obs_North2=(my_obs%16)/8;
			obs_East2=((my_obs%16)-obs_North2*8)/4;
			obs_South2=((my_obs%16)-obs_North2*8-obs_East2*4)/2;
			obs_West2=((my_obs%16)-obs_North2*8-obs_East2*4-obs_South2*2);
			my_obs = my_obs/16;
			obs_North_East2= (my_obs%16)/8;
			obs_South_East2=((my_obs % 16)-obs_North_East2*8)/4;
			obs_South_West2=((my_obs % 16)-obs_North_East2*8-obs_South_East2*4)/2;
			obs_North_West2=((my_obs % 16)-obs_North_East2*8-obs_South_East2*4-obs_South_West2*2);
		}
		//PrintObs(state, obs,cout);
		//logi<<"Refracted as:"<< obs_North << obs_East <<obs_South<<obs_West<<endl;

		int truth_North,truth_East,truth_South,truth_West;
		int truth_NE, truth_SE, truth_SW, truth_NW;
		int truth_North2,truth_East2,truth_South2,truth_West2;
		int truth_NE2, truth_SE2, truth_SW2, truth_NW2;
		truth_North=nav_state.Grid(nav_state.rob.x, nav_state.rob.y + 1);
		truth_East=nav_state.Grid(nav_state.rob.x + 1, nav_state.rob.y );
		truth_South=nav_state.Grid(nav_state.rob.x,  nav_state.rob.y -1);
		truth_West=nav_state.Grid(nav_state.rob.x - 1, nav_state.rob.y );
		truth_NE=nav_state.Grid(nav_state.rob.x + 1,nav_state.rob.y +1);
		truth_SE=nav_state.Grid(nav_state.rob.x + 1,nav_state.rob.y -1);
		truth_SW=nav_state.Grid(nav_state.rob.x - 1,nav_state.rob.y -1);
		truth_NW=nav_state.Grid(nav_state.rob.x - 1,nav_state.rob.y +1);
		if(num_obs_bits > 8)
		{
			truth_North2=nav_state.Grid(nav_state.rob.x, nav_state.rob.y + 2);
			truth_East2=nav_state.Grid(nav_state.rob.x + 2, nav_state.rob.y );
			truth_South2=nav_state.Grid(nav_state.rob.x,  nav_state.rob.y -2);
			truth_West2=nav_state.Grid(nav_state.rob.x - 2, nav_state.rob.y );
			truth_NE2=nav_state.Grid(nav_state.rob.x + 2,nav_state.rob.y +2);
			truth_SE2=nav_state.Grid(nav_state.rob.x + 2,nav_state.rob.y -2);
			truth_SW2=nav_state.Grid(nav_state.rob.x - 2,nav_state.rob.y -2);
			truth_NW2=nav_state.Grid(nav_state.rob.x - 2,nav_state.rob.y +2);
		}
		float Noise=OBS_NOISE;
		if(obs_North==truth_North)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: N"<<endl;
		}
		if(obs_East==truth_East)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: E"<<endl;
		}
		if(obs_South==truth_South)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: S"<<endl;
		}
		if(obs_West==truth_West)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: W"<<endl;
		}
		if(obs_North_East==truth_NE)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: NE"<<endl;
		}
		if(obs_South_East==truth_SE)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise:SE"<<endl;
		}
		if(obs_South_West==truth_SW)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: SW"<<endl;
		}
		if(obs_North_West==truth_NW)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: NW"<<endl;
		}
		if(num_obs_bits > 8)
		{

			prob = prob*100000; //As we are calculating likelihood, multiplying probability with a constant so that it does not become very small
			if(obs_North2==truth_North2)
				prob*=1-Noise;
			else{
				prob*=Noise;
				//if(DESPOT::Debug_mode)
				//	cout<<"Obs noise: N"<<endl;
			}
			if(obs_East2==truth_East2)
				prob*=1-Noise;
			else{
				prob*=Noise;
				//if(DESPOT::Debug_mode)
				//	cout<<"Obs noise: E"<<endl;
			}
			if(obs_South2==truth_South2)
				prob*=1-Noise;
			else{
				prob*=Noise;
				//if(DESPOT::Debug_mode)
				//	cout<<"Obs noise: S"<<endl;
			}
			if(obs_West2==truth_West2)
				prob*=1-Noise;
			else{
				prob*=Noise;
				//if(DESPOT::Debug_mode)
				//	cout<<"Obs noise: W"<<endl;
			}
			if(obs_North_East2==truth_NE2)
				prob*=1-Noise;
			else{
				prob*=Noise;
				//if(DESPOT::Debug_mode)
				//	cout<<"Obs noise: NE"<<endl;
			}
			if(obs_South_East2==truth_SE2)
				prob*=1-Noise;
			else{
				prob*=Noise;
				//if(DESPOT::Debug_mode)
				//	cout<<"Obs noise:SE"<<endl;
			}
			if(obs_South_West2==truth_SW2)
				prob*=1-Noise;
			else{
				prob*=Noise;
				//if(DESPOT::Debug_mode)
				//	cout<<"Obs noise: SW"<<endl;
			}
			if(obs_North_West2==truth_NW2)
				prob*=1-Noise;
			else{
				prob*=Noise;
				//if(DESPOT::Debug_mode)
				//	cout<<"Obs noise: NW"<<endl;
			}
		}
		return prob;
}

DEVICE Dvc_State* Dvc_UncNavigation::Allocate(int state_id, double weight) const {
	//Dvc_UncNavigationState* state = Dvc_memory_pool_.Allocate();
	Dvc_UncNavigationState* state = new Dvc_UncNavigationState();
	state->state_id = state_id;
	state->weight = weight;

	return state;
}

DEVICE Dvc_State* Dvc_UncNavigation::Dvc_Get(Dvc_State* particles, int pos) {
	Dvc_UncNavigationState* particle_i= static_cast<Dvc_UncNavigationState*>(particles)+pos;

	return particle_i;
}

DEVICE Dvc_State* Dvc_UncNavigation::Dvc_Alloc( int num) {
	//Dvc_UncNavigationState* state = Dvc_memory_pool_.Allocate();
	Dvc_UncNavigationState* state = (Dvc_UncNavigationState*)malloc(num*sizeof(Dvc_UncNavigationState));

	for(int i=0;i<num;i++)
		state[i].SetAllocated();
	return state;
}

DEVICE Dvc_State* Dvc_UncNavigation::Dvc_Copy(const Dvc_State* particles, int pos) {
	//Dvc_UncNavigationState* state = Dvc_memory_pool_.Allocate();
	const Dvc_UncNavigationState* particle_i= static_cast<const Dvc_UncNavigationState*>(particles)+pos;
	Dvc_UncNavigationState* state = new Dvc_UncNavigationState();

	*state = *particle_i;
	state->SetAllocated();
	return state;
}
DEVICE void Dvc_UncNavigation::Dvc_Copy_NoAlloc(Dvc_State* des, const Dvc_State* src, int pos, bool offset_des) {
	/*Pass member values, assign member pointers to existing state pointer*/
	const Dvc_UncNavigationState* src_i= static_cast<const Dvc_UncNavigationState*>(src)+pos;
	if(!offset_des) pos=0;
	Dvc_UncNavigationState* des_i= static_cast<const Dvc_UncNavigationState*>(des)+pos;

	*des_i = *src_i;
	des_i->SetAllocated();
}

DEVICE void Dvc_UncNavigation::Dvc_Free(Dvc_State* particle) {
	delete static_cast<Dvc_UncNavigationState*>(particle);
}

DEVICE int Dvc_UncNavigation::Dvc_NumObservations() { // one dummy terminal state
	return (int)pow(2.0, 1.0*num_obs_bits);
}


} // namespace despot
