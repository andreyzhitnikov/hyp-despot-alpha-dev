#include "hip/hip_runtime.h"
#include "GPU_UncNavigation.h"

#include "GPU_base_unc_navigation.h"
#include <base_unc_navigation.h>
#include <despot/GPUutil/GPUrandom.h>

using namespace std;

namespace despot {

DEVICE Dvc_UncNavigation::Dvc_UncNavigation()

{

}


DEVICE Dvc_UncNavigation::~Dvc_UncNavigation()
{
}

DEVICE bool Dvc_UncNavigation::Dvc_Step(Dvc_State& state, float rand_num, int action, float& reward,
	OBS_TYPE& obs) {

	Dvc_UncNavigationState& nav_state = static_cast<Dvc_UncNavigationState&>(state);//copy contents, link cells to existing ones
	bool terminal=false;
	reward = 0;

	int dir=threadIdx.y;

	if(dir==0)
	{
		terminal=(nav_state.rob==nav_state.goal);

		reward=-0.1;// small cost for one step
		DvcCoord rob_pos=nav_state.rob;

		float prob=1.0f-STEP_NOISE;

		if (action < E_STAY && terminal!=true) { // Move
			// only succeed with 80% chance
			rob_pos +=(rand_num<prob)? Dvc_Compass::GetDirections(action):DvcCoord(0,0);
			bool validmove=(nav_state.Inside(rob_pos) && nav_state.CollisionCheck(rob_pos)==false);

			nav_state.rob=validmove?rob_pos:nav_state.rob;
			reward=validmove?-0.1:-1;
			reward=(nav_state.rob==nav_state.goal)?/*10*/GOAL_REWARD:reward;
		}

		if (action == E_STAY) { // Sample
			reward=-0.2;
		}

		obs=0;//Initialize obs
	}

	OBS_TYPE obs_i=0;

	unsigned long long int Temp=INIT_QUICKRANDSEED;
	for(dir=0;dir<8;dir++)
	{
		switch(dir)
		{
		case 3:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x,nav_state.rob.y+1):!nav_state.Grid(nav_state.rob.x,nav_state.rob.y+1);
			break;
		case 2:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x+1,nav_state.rob.y):!nav_state.Grid(nav_state.rob.x+1,nav_state.rob.y);
			break;
		case 1:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x,nav_state.rob.y-1):!nav_state.Grid(nav_state.rob.x,nav_state.rob.y-1);
			break;
		case 0:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x-1,nav_state.rob.y):!nav_state.Grid(nav_state.rob.x-1,nav_state.rob.y);
			break;
		case 4:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x-1,nav_state.rob.y+1):!nav_state.Grid(nav_state.rob.x-1,nav_state.rob.y+1);
			break;
		case 5:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x-1,nav_state.rob.y-1):!nav_state.Grid(nav_state.rob.x-1,nav_state.rob.y-1);
			break;
		case 6:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x+1,nav_state.rob.y-1):!nav_state.Grid(nav_state.rob.x+1,nav_state.rob.y-1);
			break;
		case 7:
			rand_num=Dvc_QuickRandom::RandGeneration(&Temp, rand_num);
			obs_i=(rand_num<1-OBS_NOISE)?nav_state.Grid(nav_state.rob.x+1,nav_state.rob.y+1):!nav_state.Grid(nav_state.rob.x+1,nav_state.rob.y+1);
			break;
		}
		obs=(obs|(obs_i<<dir));
	}

	if(obs>=Dvc_NumObservations())
		printf("Wrong obs %d", obs);

	if(threadIdx.y==0)
	{
		if(terminal){reward=0;obs=Dvc_NumObservations()-1;}
	}
	return terminal;
}

DEVICE float Dvc_UncNavigation::Dvc_ObsProb(OBS_TYPE& obs, Dvc_State& state, int action)
{
		float prob=1;
		Dvc_UncNavigationState nav_state(static_cast<const Dvc_UncNavigationState&>(state));

		int obs_North=(obs%16)/8;
		int obs_East=((obs%16)-obs_North*8)/4;
		int obs_South=((obs%16)-obs_North*8-obs_East*4)/2;
		int obs_West=((obs%16)-obs_North*8-obs_East*4-obs_South*2);
		int obs_North_East=obs/(int)pow(2.0,7.0);
		int obs_South_East=(obs-obs_North_East*(int)pow(2.0,7.0))/(int)pow(2.0,6.0);
		int obs_South_West=(obs-obs_North_East*(int)pow(2.0,7.0)-obs_South_East*(int)pow(2.0,6.0))/(int)pow(2.0,5.0);
		int obs_North_West=(obs-obs_North_East*(int)pow(2.0,7.0)-obs_South_East*(int)pow(2.0,6.0)-obs_South_West*(int)pow(2.0,5.0))/(int)pow(2.0,4.0);

		//PrintObs(state, obs,cout);
		//logi<<"Refracted as:"<< obs_North << obs_East <<obs_South<<obs_West<<endl;

		int truth_North,truth_East,truth_South,truth_West;
		int truth_NE, truth_SE, truth_SW, truth_NW;
		truth_North=nav_state.Grid(nav_state.rob.x, nav_state.rob.y + 1);
		truth_East=nav_state.Grid(nav_state.rob.x + 1, nav_state.rob.y );
		truth_South=nav_state.Grid(nav_state.rob.x,  nav_state.rob.y -1);
		truth_West=nav_state.Grid(nav_state.rob.x - 1, nav_state.rob.y );
		truth_NE=nav_state.Grid(nav_state.rob.x + 1,nav_state.rob.y +1);
		truth_SE=nav_state.Grid(nav_state.rob.x + 1,nav_state.rob.y -1);
		truth_SW=nav_state.Grid(nav_state.rob.x - 1,nav_state.rob.y -1);
		truth_NW=nav_state.Grid(nav_state.rob.x - 1,nav_state.rob.y +1);

		float Noise=OBS_NOISE;
		if(obs_North==truth_North)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: N"<<endl;
		}
		if(obs_East==truth_East)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: E"<<endl;
		}
		if(obs_South==truth_South)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: S"<<endl;
		}
		if(obs_West==truth_West)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: W"<<endl;
		}
		if(obs_North_East==truth_NE)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: NE"<<endl;
		}
		if(obs_South_East==truth_SE)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise:SE"<<endl;
		}
		if(obs_South_West==truth_SW)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: SW"<<endl;
		}
		if(obs_North_West==truth_NW)
			prob*=1-Noise;
		else{
			prob*=Noise;
			//if(DESPOT::Debug_mode)
			//	cout<<"Obs noise: NW"<<endl;
		}
		return prob;
}

DEVICE Dvc_State* Dvc_UncNavigation::Allocate(int state_id, double weight) const {
	//Dvc_UncNavigationState* state = Dvc_memory_pool_.Allocate();
	Dvc_UncNavigationState* state = new Dvc_UncNavigationState();
	state->state_id = state_id;
	state->weight = weight;

	return state;
}

DEVICE Dvc_State* Dvc_UncNavigation::Dvc_Get(Dvc_State* particles, int pos) {
	Dvc_UncNavigationState* particle_i= static_cast<Dvc_UncNavigationState*>(particles)+pos;

	return particle_i;
}

DEVICE Dvc_State* Dvc_UncNavigation::Dvc_Alloc( int num) {
	//Dvc_UncNavigationState* state = Dvc_memory_pool_.Allocate();
	Dvc_UncNavigationState* state = (Dvc_UncNavigationState*)malloc(num*sizeof(Dvc_UncNavigationState));

	for(int i=0;i<num;i++)
		state[i].SetAllocated();
	return state;
}

DEVICE Dvc_State* Dvc_UncNavigation::Dvc_Copy(const Dvc_State* particles, int pos) {
	//Dvc_UncNavigationState* state = Dvc_memory_pool_.Allocate();
	const Dvc_UncNavigationState* particle_i= static_cast<const Dvc_UncNavigationState*>(particles)+pos;
	Dvc_UncNavigationState* state = new Dvc_UncNavigationState();

	*state = *particle_i;
	state->SetAllocated();
	return state;
}
DEVICE void Dvc_UncNavigation::Dvc_Copy_NoAlloc(Dvc_State* des, const Dvc_State* src, int pos, bool offset_des) {
	/*Pass member values, assign member pointers to existing state pointer*/
	const Dvc_UncNavigationState* src_i= static_cast<const Dvc_UncNavigationState*>(src)+pos;
	if(!offset_des) pos=0;
	Dvc_UncNavigationState* des_i= static_cast<const Dvc_UncNavigationState*>(des)+pos;

	*des_i = *src_i;
	des_i->SetAllocated();
}

DEVICE void Dvc_UncNavigation::Dvc_Free(Dvc_State* particle) {
	delete static_cast<Dvc_UncNavigationState*>(particle);
}

DEVICE int Dvc_UncNavigation::Dvc_NumObservations() { // one dummy terminal state
	return 256;
}


} // namespace despot
