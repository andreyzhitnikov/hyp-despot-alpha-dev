#include <despot/core/solver.h>
#include <despot/interface/belief.h>
#include <despot/interface/world.h>
#include <despot/logger.h>
#include <despot/planner.h>
#include <despot/GPUcore/thread_globals.h>
#include <string.h>
#include <despot/GPUcore/disabled_util.h>
#include <iostream>
#include <cstring>
#include <despot/GPUconfig.h>
#include <despot/GPUutil/GPUrandom.h>


using namespace std;
namespace despot {

static int cudaCoreNum = 0;
static int asyncEngineCount = 0;

int getSPcores(hipDeviceProp_t devProp) {
	int cores = 0;
	int mp = devProp.multiProcessorCount;
	switch (devProp.major) {
	case 2: // Fermi
		if (devProp.minor == 1)
			cores = mp * 48;
		else
			cores = mp * 32;
		break;
	case 3: // Kepler
		cores = mp * 192;
		break;
	case 5: // Maxwell
		cores = mp * 128;
		break;
	case 6: // Pascal
		if (devProp.minor == 1)
			cores = mp * 128;
		else if (devProp.minor == 0)
			cores = mp * 64;
		else
			printf("Unknown device type\n");
		break;
	default:
		printf("Unknown device type\n");
		break;
	}
	return cores;
}

void SetupGPU() {
	int devicesCount;
	hipGetDeviceCount(&devicesCount);
	int deviceIndex = Globals::config.GPUid;

	hipSetDevice(deviceIndex);
	hipGetDevice(&deviceIndex);

	hipDeviceProp_t deviceProperties;
	hipGetDeviceProperties(&deviceProperties, deviceIndex);
	if (deviceProperties.major >= 2 && deviceProperties.minor >= 0) {
		cout << "Device:" << "(" << deviceIndex << ")" << deviceProperties.name
				<< endl;
		cout << "Multi-processors:" << deviceProperties.multiProcessorCount
				<< endl;
		size_t heapsize;
		hipDeviceGetLimit(&heapsize, hipLimitMallocHeapSize);
		hipDeviceSetLimit(hipLimitMallocHeapSize, heapsize * 10);
		hipDeviceGetLimit(&heapsize, hipLimitMallocHeapSize);
		cudaCoreNum = getSPcores(deviceProperties);
		cout << "Number of cores:" << cudaCoreNum << endl;

		asyncEngineCount = deviceProperties.asyncEngineCount;
		cout << "Number of asynchronous engines:" << asyncEngineCount << endl;

		if (asyncEngineCount >= 2) {
			Globals::SetupCUDAStreams();
		} else
			cout << "The current GPU no enough asyncEngine (<2)" << endl;
	}
	std::memset((void*)&sa, 0, sizeof(struct sigaction));
	sigemptyset(&sa.sa_mask);
	sa.sa_sigaction = segfault_sigaction;
	sa.sa_flags   = SA_SIGINFO;

	sigaction(SIGSEGV, &sa, NULL);

	cout << "GPU setup done." << endl;

}


void Planner::PrepareGPU() {
	if (Globals::config.useGPU) {
		SetupGPU();

		//Setup global configurations in GPU and randnum generators

		Dvc_Config::CopyToGPU(&config);

		Dvc_Random::init(Globals::config.num_scenarios);

		Dvc_QuickRandom::InitRandGen();
	}
}


void Planner::ClearGPU() {

	Dvc_Config::Clear();

	Dvc_Random::clear();

	Dvc_QuickRandom::DestroyRandGen();
}

} //namespace despot