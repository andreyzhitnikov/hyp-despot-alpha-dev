#include "hip/hip_runtime.h"
#include <despot/solver/Hyp_despot.h>
#include <despot/core/globals.h>

#include <despot/GPUcore/thread_globals.h>
#include <despot/GPUcore/disabled_util.h>

#include <despot/GPUinterface/GPUupper_bound.h>
#include <despot/GPUinterface/GPUlower_bound.h>
#include <despot/GPUinterface/GPUdefault_policy.h>
#include <despot/GPUinterface/GPUpolicy_graph.h>

#include <despot/planner.h>
#include <string.h>

using namespace std;

// atomic time trackers for multi-threaded time recording (time used by all threads are accumulated)

static atomic<double> InitBoundTime(0);
static atomic<double> AveRewardTime(0);
static atomic<double> MakeObsNodeTime(0);
static atomic<double> CopyParticleTime(0);
static atomic<double> CopyHistoryTime(0);
static atomic<double> MakePartitionTime(0);
static atomic<int> HitCount(0);
static atomic<double> AveNumParticles(0);
static atomic<double> TotalExpansionTime(0);
static atomic<double> DataBackTime(0);



// Observation type 

namespace despot {
OBS_PARAM Obs_type= OBS_LONG64;

/**
 * In HyP-DESPOT, all internal data used by GPU are preallocated as arrays.
 * For example, reward array, particle array, ub array, lb array etc.
 * Memory for these arrays are further arranged in contigous memory defined as Hst_MC_Data (in host memory) and Dvc_MC_Data (in device memory).
 * This is to optimize the data copying speed between CPU and GPU.
 * A seperate list is maintained for each of the CPU expansion threads
*/

static Dvc_RandomStreams** Dvc_streams = NULL;
static Dvc_History** Dvc_history = NULL;


static int MC_DataSize=0;
static void** Dvc_MC_Data = NULL;
static void** Hst_MC_Data = NULL;

static float** Dvc_r_all_a = NULL;
static float** Hst_r_all_a = NULL;

static float** Dvc_r_all_a_and_p = NULL; //used by despot with alpha vector update
static float** Hst_r_all_a_and_p = NULL; //used by despot with alpha vector update

static float ** Dvc_ub_all_a_p = NULL;
static float ** Hst_ub_all_a_p = NULL;

static float** Dvc_uub_all_a_p = NULL;
static float** Hst_uub_all_a_p = NULL;

static Dvc_ValuedAction** Dvc_lb_all_a_p = NULL;
static Dvc_ValuedAction** Hst_lb_all_a_p = NULL;


static OBS_TYPE** Dvc_obs_all_a_and_p=NULL;
static OBS_TYPE** Hst_obs_all_a_and_p=NULL;

static bool** Dvc_term_all_a_and_p=NULL;
static bool** Hst_term_all_a_and_p=NULL;

static int** Dvc_obs_int_all_a_and_p=NULL;
static int** Hst_obs_int_all_a_and_p=NULL;

static float ** Dvc_obs_prob_all_a_p_obs = NULL; //used by despot with alpha vector update
static float ** Hst_obs_prob_all_a_p_obs = NULL; //used by despot with alpha vector update

/**
 * Dvc_particleIDs_long: pre-allocated device memory to hold the IDs of particles for expansion and rollout.
 * A seperate list is maintained for each of the CPU expansion threads
*/
static int** Dvc_particleIDs_long = NULL;

/**
 * Dvc_stepped_particles_all_a: pre-allocated device memory to hold the stepped particles corresponding to all expansion actions.
 * A seperate list is maintained for each of the CPU expansion threads
*/
static Dvc_State** Dvc_stepped_particles_all_a=NULL;


void PrintThreadData(int ThreadID);
void initGPUHistory();
void clearGPUHistory();

void DESPOT::PrepareGPUMemory(const DSPOMDP* model, int num_actions,
		int num_obs) {
	clock_t start = clock();

	int num_copies =1;
	if(Globals::config.use_multi_thread_)num_copies=Globals::config.NUM_THREADS;

	if (Globals::config.NUM_THREADS > 1 && Globals::config.use_multi_thread_) {
		Dvc_streams = new Dvc_RandomStreams*[Globals::config.NUM_THREADS];

		if(Globals::config.track_alpha_vector)
		{
			Dvc_r_all_a_and_p = new float*[Globals::config.NUM_THREADS]; //used by despot with alpha vector update
			Hst_r_all_a_and_p = new float*[Globals::config.NUM_THREADS]; //used by despot with alpha vector update
			Dvc_obs_prob_all_a_p_obs = new float*[Globals::config.NUM_THREADS]; //used by despot with alpha vector update
			Hst_obs_prob_all_a_p_obs = new float*[Globals::config.NUM_THREADS]; //used by despot with alpha vector update
		}
		else
		{
			Dvc_r_all_a = new float*[Globals::config.NUM_THREADS];
			Hst_r_all_a = new float*[Globals::config.NUM_THREADS];
		}
		Dvc_obs_all_a_and_p = new OBS_TYPE*[Globals::config.NUM_THREADS];
		Hst_obs_all_a_and_p = new OBS_TYPE*[Globals::config.NUM_THREADS];
		Dvc_obs_int_all_a_and_p = new int*[Globals::config.NUM_THREADS];
		Hst_obs_int_all_a_and_p = new int*[Globals::config.NUM_THREADS];
		Dvc_term_all_a_and_p = new bool*[Globals::config.NUM_THREADS];
		Hst_term_all_a_and_p = new bool*[Globals::config.NUM_THREADS];
		Dvc_ub_all_a_p = new float*[Globals::config.NUM_THREADS];
		Dvc_uub_all_a_p = new float*[Globals::config.NUM_THREADS];
		Dvc_lb_all_a_p = new Dvc_ValuedAction*[Globals::config.NUM_THREADS];
		Hst_lb_all_a_p = new Dvc_ValuedAction*[Globals::config.NUM_THREADS];
		Hst_ub_all_a_p = new float*[Globals::config.NUM_THREADS];
		Hst_uub_all_a_p = new float*[Globals::config.NUM_THREADS];
		Dvc_particleIDs_long = new int*[Globals::config.NUM_THREADS];
		Dvc_MC_Data=new void*[Globals::config.NUM_THREADS];
		Hst_MC_Data=new void*[Globals::config.NUM_THREADS];
	} else {
		num_copies = 1;
		Dvc_streams = new Dvc_RandomStreams*;

		if(Globals::config.track_alpha_vector)
		{
			Dvc_r_all_a_and_p = new float*; //used by despot with alpha vector update
			Hst_r_all_a_and_p = new float*; //used by despot with alpha vector update
			Dvc_obs_prob_all_a_p_obs = new float*; //used by despot with alpha vector update
			Hst_obs_prob_all_a_p_obs = new float*; //used by despot with alpha vector update
		}
		else
		{
			Dvc_r_all_a = new float*;
			Hst_r_all_a = new float*;
		}
		Dvc_obs_all_a_and_p = new OBS_TYPE*;
		Hst_obs_all_a_and_p = new OBS_TYPE*;
		Dvc_obs_int_all_a_and_p = new int*;
		Hst_obs_int_all_a_and_p = new int*;
		Dvc_term_all_a_and_p = new bool*;
		Hst_term_all_a_and_p = new bool*;
		Dvc_ub_all_a_p = new float*;
		Dvc_uub_all_a_p = new float*;
		Dvc_lb_all_a_p = new Dvc_ValuedAction*;
		Hst_lb_all_a_p = new Dvc_ValuedAction*;
		Hst_ub_all_a_p = new float*;
		Hst_uub_all_a_p = new float*;
		Dvc_particleIDs_long = new int*;
		Dvc_MC_Data=new void*;
		Hst_MC_Data=new void*;
	}

	for (int i = 0; i < num_copies; i++) {
		HANDLE_ERROR(
				hipMallocManaged((void** )&Dvc_streams[i],
						sizeof(Dvc_RandomStreams)));
		Dvc_RandomStreams::Init(Dvc_streams[i], Globals::config.num_scenarios,
				config.search_depth,(i==0)?true:false);
		int offset_obs=0;int offset_term=0;

		offset_obs=num_actions * sizeof(float);
		if(Globals::config.track_alpha_vector)
		{
			offset_obs=Globals::config.num_scenarios * num_actions * sizeof(float);
		}
		if(Obs_type==OBS_INT_ARRAY)
		{

			int blocksize=sizeof(int)*num_Obs_element_in_GPU;
			if(offset_obs%blocksize!=0)
				offset_obs=(offset_obs/blocksize+1)*blocksize;

			offset_term=offset_obs+num_actions * Globals::config.num_scenarios * blocksize;
			if(offset_term%sizeof(bool)!=0) offset_term=(offset_term/sizeof(bool)+1)*sizeof(bool);
		}
		else
		{

			if(offset_obs%sizeof(OBS_TYPE)!=0) offset_obs=(offset_obs/sizeof(OBS_TYPE)+1)*sizeof(OBS_TYPE);

			offset_term=offset_obs+num_actions * Globals::config.num_scenarios * sizeof(OBS_TYPE);
			if(offset_term%sizeof(bool)!=0) offset_term=(offset_term/sizeof(bool)+1)*sizeof(bool);
		}

		int offset_ub=offset_term+num_actions * Globals::config.num_scenarios * sizeof(bool);
		if(offset_ub%sizeof(float)!=0) offset_ub=(offset_ub/sizeof(float)+1)*sizeof(float);

		int offset_uub=offset_ub+num_actions * Globals::config.num_scenarios * sizeof(float);
		if(offset_uub%sizeof(float)!=0) offset_uub=(offset_uub/sizeof(float)+1)*sizeof(float);

		int offset_lb=offset_uub+num_actions * Globals::config.num_scenarios * sizeof(float);
		if(offset_lb%sizeof(Dvc_ValuedAction)!=0) offset_lb=(offset_lb/sizeof(Dvc_ValuedAction)+1)*sizeof(Dvc_ValuedAction);

		int offset_obs_prob = 0;
		if(Globals::config.track_alpha_vector)
		{
			offset_obs_prob = offset_lb+num_actions * Globals::config.num_scenarios * sizeof(Dvc_ValuedAction);
			if(offset_obs_prob%sizeof(float)!=0) offset_obs_prob=(offset_obs_prob/sizeof(float)+1)*sizeof(float);

			MC_DataSize=offset_obs_prob+num_actions * Globals::config.num_scenarios * Globals::config.num_scenarios * sizeof(float);

		}
		else
		{

			MC_DataSize=offset_lb+num_actions * Globals::config.num_scenarios * sizeof(Dvc_ValuedAction);
		}
		HANDLE_ERROR(
				hipMalloc((void** )&Dvc_MC_Data[i],MC_DataSize));
		HANDLE_ERROR(
				hipHostAlloc((void** )&Hst_MC_Data[i],MC_DataSize	, 0));
		if(Globals::config.track_alpha_vector)
		{
			Dvc_r_all_a_and_p[i]=(float*)Dvc_MC_Data[i];
			Hst_r_all_a_and_p[i]=(float*)Hst_MC_Data[i];
		}
		else
		{
			Dvc_r_all_a[i]=(float*)Dvc_MC_Data[i];
			Hst_r_all_a[i]=(float*)Hst_MC_Data[i];
		}

		if(Obs_type==OBS_INT_ARRAY)
		{
			Dvc_obs_int_all_a_and_p[i]=(int*)(Dvc_MC_Data[i]+offset_obs);
			Hst_obs_int_all_a_and_p[i]=(int*)(Hst_MC_Data[i]+offset_obs);
		}
		else{
			Dvc_obs_all_a_and_p[i]=(OBS_TYPE*)(Dvc_MC_Data[i]+offset_obs);
			Hst_obs_all_a_and_p[i]=(OBS_TYPE*)(Hst_MC_Data[i]+offset_obs);
		}

		Dvc_term_all_a_and_p[i]=(bool*)(Dvc_MC_Data[i]+offset_term);
		Hst_term_all_a_and_p[i]=(bool*)(Hst_MC_Data[i]+offset_term);

		Dvc_ub_all_a_p[i]=(float*)(Dvc_MC_Data[i]+offset_ub);
		Hst_ub_all_a_p[i]=(float*)(Hst_MC_Data[i]+offset_ub);

		Dvc_uub_all_a_p[i]=(float*)(Dvc_MC_Data[i]+offset_uub);
		Hst_uub_all_a_p[i]=(float*)(Hst_MC_Data[i]+offset_uub);

		Dvc_lb_all_a_p[i]=(Dvc_ValuedAction*)(Dvc_MC_Data[i]+offset_lb);
		Hst_lb_all_a_p[i]=(Dvc_ValuedAction*)(Hst_MC_Data[i]+offset_lb);

		if(Globals::config.track_alpha_vector)
		{
			Dvc_obs_prob_all_a_p_obs[i] = (float*)(Dvc_MC_Data[i]+offset_obs_prob);
			Hst_obs_prob_all_a_p_obs[i] = (float*)(Hst_MC_Data[i]+offset_obs_prob);
		}

		if(Globals::config.track_alpha_vector)
		{
			HANDLE_ERROR(hipMalloc((void** )&Dvc_particleIDs_long[i],
									Globals::config.num_scenarios *num_actions* sizeof(int)));
		}
		else
		{
			HANDLE_ERROR(
				hipMalloc((void** )&Dvc_particleIDs_long[i],
						Globals::config.num_scenarios * sizeof(int)));
		}
		cout<<"Dvc_particleIDs_long[i]: "<< Dvc_particleIDs_long[i] << ",Globals::config.num_scenarios="<< Globals::config.num_scenarios<<endl;

	}

	cout<<"GPUDespot ouput Data size: "<<MC_DataSize<<"*"<<num_copies<<" bytes"<<endl;

	model->AllocGPUParticles(Globals::config.num_scenarios, MEMORY_MODE(INIT), &Dvc_stepped_particles_all_a);
	initGPUHistory();

	cout << "GPU memory init time:"
			<< (double) (clock() - start) / CLOCKS_PER_SEC << endl;

}


void DESPOT::ClearGPUMemory(const DSPOMDP* model) {
	int thread_count = 1;
	if (Globals::config.use_multi_thread_)
		thread_count = Globals::config.NUM_THREADS;

	for (int i = 0; i < thread_count; i++) {
		if (Dvc_streams[i] != NULL) {
			Dvc_RandomStreams::Clear(Dvc_streams[i]);
			HANDLE_ERROR(hipFree(Dvc_streams[i]));
			Dvc_streams[i] = NULL;
		}
		if (Dvc_MC_Data[i] != NULL) {
			HANDLE_ERROR(hipFree(Dvc_MC_Data[i]));
			Dvc_MC_Data[i] = NULL;
		}

		if (Hst_MC_Data[i] != NULL) {
			HANDLE_ERROR(hipHostFree(Hst_MC_Data[i]));
			Hst_MC_Data[i] = NULL;
		}

		if (Dvc_particleIDs_long[i] != NULL) {
			HANDLE_ERROR(hipFree(Dvc_particleIDs_long[i]));
			Dvc_particleIDs_long[i] = NULL;
		}
	}

	if (Globals::config.NUM_THREADS > 1 && Globals::config.use_multi_thread_) {
		delete[] Dvc_streams;
		delete[] Dvc_r_all_a;
		delete[] Hst_r_all_a;
		delete[] Dvc_obs_all_a_and_p;
		delete[] Hst_obs_all_a_and_p;
		delete[] Dvc_obs_int_all_a_and_p;
		delete[] Hst_obs_int_all_a_and_p;
		delete[] Dvc_term_all_a_and_p;
		delete[] Hst_term_all_a_and_p;
		delete[] Dvc_ub_all_a_p;
		delete[] Dvc_uub_all_a_p;
		delete[] Hst_ub_all_a_p;
		delete[] Hst_uub_all_a_p;
		delete[] Dvc_lb_all_a_p;
		delete[] Hst_lb_all_a_p;
		delete[] Dvc_particleIDs_long;
		delete[] Dvc_MC_Data;
		delete[] Hst_MC_Data;
	} else {
		delete Dvc_streams;
		delete Dvc_r_all_a;
		delete Hst_r_all_a;
		delete Dvc_obs_all_a_and_p;
		delete Hst_obs_all_a_and_p;
		delete Dvc_obs_int_all_a_and_p;
		delete Hst_obs_int_all_a_and_p;
		delete Dvc_term_all_a_and_p;
		delete Hst_term_all_a_and_p;
		delete Dvc_ub_all_a_p;
		delete Dvc_uub_all_a_p;
		delete Hst_ub_all_a_p;
		delete Hst_uub_all_a_p;
		delete Dvc_lb_all_a_p;
		delete Hst_lb_all_a_p;
		delete Dvc_particleIDs_long;
		delete Dvc_MC_Data;
		delete Hst_MC_Data;
	}

	Globals::DestroyCUDAStreams();
	model->DeleteGPUParticles(MEMORY_MODE(DESTROY), Dvc_stepped_particles_all_a);
	clearGPUHistory();

}

__global__ void ShareStreamData(Dvc_RandomStreams* des,
		Dvc_RandomStreams* src) {

	des->num_streams_ = src->num_streams_;
	des->length_ = src->length_;
	for (int i = 0; i < des->num_streams_; i++) {
		des->streams_[i] = src->streams_[i];
	}
	des->position_=0;
}

void DESPOT::PrepareGPUStreams(const RandomStreams& streams) {
	clock_t start = clock();

	Dvc_RandomStreams::CopyToGPU(Dvc_streams[0], &streams);

	if (Globals::config.use_multi_thread_) {
		for (int i = 1; i < Globals::config.NUM_THREADS; i++) {
			dim3 grid1(1, 1);
			dim3 threads1(1, 1);
			ShareStreamData<<<grid1, threads1, 0, Globals::GetThreadCUDAStream(i)>>>(
					Dvc_streams[i], Dvc_streams[0]);
		}
	} else {
	}
	HANDLE_ERROR(hipDeviceSynchronize());

}





/**
 * Calculate exploration bonuses for v-nodes (applied on weu) and q-nodes (applied on ub)
 */

float DESPOT::CalExplorationValue(int depth) {
	return Globals::config.exploration_constant_o * Initial_root_gap;
}

void DESPOT::CalExplorationValue(Shared_QNode* node) {
	if(Globals::config.exploration_constant>0)
	{
		node->exploration_bonus= Globals::config.exploration_constant *
			sqrt(log(static_cast<Shared_VNode*>(((QNode*)node)->parent())->visit_count_*
					max(((QNode*)node)->parent()->Weight()*
					Globals::config.num_scenarios,1.1))
			/(node->visit_count_*max(((QNode*)node)->Weight()*
					Globals::config.num_scenarios,1.1)));

		node->exploration_bonus*=((QNode*)node)->Weight();
	}
}

void DESPOT::CalExplorationValue(Shared_VNode* node) {
	node->exploration_bonus= Globals::config.exploration_constant *
		sqrt(log(static_cast<Shared_QNode*>(((VNode*)node)->parent())->visit_count_*
				max(((VNode*)node)->parent()->Weight()*
				Globals::config.num_scenarios,1.0))
		/(node->visit_count_*max(((VNode*)node)->Weight()*
				Globals::config.num_scenarios,1.0)));

}

/**
 * Shared memory for thread blocks to hold a local copy of particle for each thread in the block
 */
extern __shared__ int localParticles[];


__global__ void DebugSteppedParticles(Dvc_State* stepped_particles, int stepped_particles_size)
{
	//int action = blockIdx.x;
	//int scenario_id = (blockIdx.y * blockDim.x + threadIdx.x) % total_num_scenarios;
	//int global_list_pos = action*total_num_scenarios + scenario_id;
	for(int i = 0; i < stepped_particles_size; i++)
	{
		printf("Checking particle %d ", i);
		Dvc_State* temp = DvcModelGet_(stepped_particles, i);
		DvcModelCopyNoAlloc_(temp, temp, 0, false);
	}

}

void DESPOT::CheckSteppedParticles(int num_actions)
{
	int stepped_particles_size = num_actions*Globals::config.num_scenarios;
	std::cout << "Checking " << stepped_particles_size << " stepped particles  ..." ;
	DebugSteppedParticles<<<1,1>>>(Dvc_stepped_particles_all_a[0], stepped_particles_size);
	hipDeviceSynchronize();
	std::cout << "Done\n";
}
/**
 * PreStep kernel (Long observation type):
 * Forward the particle copied from parent node for one simulation step to keep it up-to-date
 * Y dimemsion in thread blocks are reserved for Hetergenous element-wise parallelization
 */

__global__ void
PreStep_LongObs(int total_num_scenarios, int num_particles, Dvc_State* vnode_particles,
		const int* vnode_particleIDs, Dvc_RandomStreams* streams, int parent_action) {

	if (blockIdx.y * blockDim.x + threadIdx.x < num_particles && blockIdx.x==0) {

		int action = blockIdx.x;
		int PID = (blockIdx.y * blockDim.x + threadIdx.x) % num_particles;

		/*Step the particles*/

		/*make a local copy of the particle in shared memory*/
		if (threadIdx.y == 0) {
			DvcModelCopyToShared_(
					(Dvc_State*) ((int*) localParticles + 60 * threadIdx.x),
					vnode_particles, PID % num_particles, false);
		}
		Dvc_State* current_particle = (Dvc_State*) ((int*) localParticles + 60 * threadIdx.x);
		__syncthreads();

		OBS_TYPE obs = (OBS_TYPE) (-1);
		float reward = 0;

		/*step the local particle, get obs and reward*/

		if(parent_action>=0)
		{
			DvcModelStep_(*current_particle, streams->Entry(current_particle->scenario_id, streams->position_-1),
					parent_action, reward, obs);

			if (blockIdx.y * blockDim.x + threadIdx.x < num_particles) {
				/*Record stepped particles from parent as particles in this node*/
				if (threadIdx.y == 0 && action==0) {
					Dvc_State* temp = DvcModelGet_(vnode_particles, PID % num_particles);
					DvcModelCopyNoAlloc_(temp, current_particle,0, false);
				}
			}
		}
	}
}

/**
 * Step kernel (Long observation type):
 * Perform one simulation step for all particles using all candidate actions
 * Y dimemsion in thread blocks are reserved for Hetergenous element-wise parallelization
 */

__global__ void
Step_LongObs(int total_num_scenarios, int num_particles, Dvc_State* vnode_particles,
		const int* vnode_particleIDs, float* step_reward_all_a,
		OBS_TYPE* observations_all_a_p, Dvc_State* new_particles,
		Dvc_RandomStreams* streams, bool* terminal_all_a_p) {
	if (blockIdx.y * blockDim.x + threadIdx.x < num_particles) {

		int action = blockIdx.x;
		int PID = (blockIdx.y * blockDim.x + threadIdx.x) % num_particles;

		if (blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0)
			step_reward_all_a[action] = 0;

		/*Step the particles*/
		Dvc_State* current_particle = NULL;
		int parent_PID = vnode_particleIDs[PID];

		/*make a local copy of the particle in shared memory*/
		if (threadIdx.y == 0) {
			DvcModelCopyToShared_(
					(Dvc_State*) ((int*) localParticles + 60 * threadIdx.x),
					vnode_particles, PID % num_particles, false);
		}
		current_particle = (Dvc_State*) ((int*) localParticles + 60 * threadIdx.x);
		__syncthreads();

		OBS_TYPE obs = (OBS_TYPE) (-1);
		float reward = 0;

		int terminal = DvcModelStep_(*current_particle, streams->Entry(current_particle->scenario_id),
				action, reward, obs);
		if(!Dvc_config->track_alpha_vector)
		{
			reward = reward * current_particle->weight;
		}
		/*Record stepped particles*/
		int global_list_pos = action * total_num_scenarios + parent_PID;
		if (threadIdx.y == 0) {
			Dvc_State* temp = DvcModelGet_(new_particles, global_list_pos);
			DvcModelCopyNoAlloc_(temp, current_particle, 0, false);

			/*Record all observations for CPU usage*/
			if (!terminal) {
				observations_all_a_p[global_list_pos] = obs;
			} else {
				observations_all_a_p[global_list_pos] = (OBS_TYPE) (-1);
			}

			if(!Dvc_config->track_alpha_vector)
			{
				/*Accumulate rewards of all particles from the v-node for CPU usage*/
				atomicAdd(step_reward_all_a + action, reward);
			}
			else
			{
				step_reward_all_a[global_list_pos] = reward;
			}
		}
		if (threadIdx.y == 0)
			terminal_all_a_p[global_list_pos] = terminal;
	}
}

/**
 * Update_and_Step kernel (Int array observation type):
 * Update parent particles with one step of MC simulation, then perform one simulation step for all particles using all candidate actions
 * Y dimemsion in thread blocks are reserved for Hetergenous element-wise parallelization
 */

__global__ void
Update_and_Step_IntArrayObs(int total_num_scenarios, int num_particles, Dvc_State* vnode_particles,
		const int* vnode_particleIDs, float* step_reward_all_a,
		int* observations_all_a_p,const int num_obs_elements,
		Dvc_State* new_particles,
		Dvc_RandomStreams* streams, bool* terminal_all_a_p
		, int parent_action,
		int Shared_mem_per_particle) {


	if (blockIdx.y * blockDim.x + threadIdx.x < num_particles) {
		__shared__ int Intobs[32*60];

		int action = blockIdx.x;
		int PID = (blockIdx.y * blockDim.x + threadIdx.x) % num_particles;
		int parent_PID = -1;

		if (blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0)
			step_reward_all_a[action] = 0;

		Dvc_State* current_particle = NULL;

		parent_PID = vnode_particleIDs[PID];

		/*make a local copy of the particle in shared memory*/
		if (threadIdx.y == 0) {
			DvcModelCopyToShared_(
					(Dvc_State*) ((int*) localParticles + Shared_mem_per_particle * threadIdx.x),
					vnode_particles, PID % num_particles, false);
		}
		current_particle = (Dvc_State*) ((int*) localParticles + Shared_mem_per_particle * threadIdx.x);
		__syncthreads();

		int terminal = false;
		float reward = 0;

		/*Update the local particle to the current depth using the action taken in the parent node*/
		if(parent_action>=0)
		{
			if(DvcModelStepIntObs_)
			{

				terminal = DvcModelStepIntObs_(*current_particle, streams->Entry(current_particle->scenario_id, streams->position_-1),
							parent_action, reward, Intobs+threadIdx.x*num_obs_elements);
			}
			else
			{
				printf("Undefined DvcModelStepIntObs_!\n");
			}
			__syncthreads();

			if (blockIdx.y * blockDim.y + threadIdx.y < num_particles) {
				/*Record stepped particles from parent as particles in this node*/
				if (threadIdx.y == 0 && action==0) {

					Dvc_State* temp = DvcModelGet_(vnode_particles, PID % num_particles);
					DvcModelCopyNoAlloc_(temp, current_particle,0, false);
				}
			}
			__syncthreads();

		}

		/*step the local particle, get obs and reward*/

		if(DvcModelStepIntObs_)
		{
			terminal = DvcModelStepIntObs_(*current_particle, streams->Entry(current_particle->scenario_id),
					action, reward, Intobs+threadIdx.x*num_obs_elements);
		}
		else
		{
			printf("Undefined DvcModelStepIntObs_!\n");
		}

		reward = reward * current_particle->weight;


		/*Record stepped particles*/
		int global_list_pos = action * total_num_scenarios + parent_PID;


		if (threadIdx.y == 0) {

			Dvc_State* temp = DvcModelGet_(new_particles, global_list_pos);
			DvcModelCopyNoAlloc_(temp, current_particle, 0, false);

			/*Record all observations for CPU usage*/
			if (!terminal) {
				for(int i=0;i<num_obs_elements;i++)
					observations_all_a_p[global_list_pos*num_obs_elements+i] = Intobs[threadIdx.x*num_obs_elements+i];
			} else {
				observations_all_a_p[global_list_pos*num_obs_elements] = 0;//no content in obs list
			}

			/*Accumulate rewards of all particles from the v-node for CPU usage*/
			atomicAdd(step_reward_all_a + action, reward);

			if (threadIdx.y == 0)
				terminal_all_a_p[global_list_pos] = terminal;
		}
	}
}
__global__ void
PreStep_IntObs(int num_particles, Dvc_State* vnode_particles,
		const int num_obs_elements,
		Dvc_RandomStreams* streams, int parent_action,
		int Shared_mem_per_particle) {

	if (blockIdx.y * blockDim.x + threadIdx.x < num_particles) {
		__shared__ int Intobs[32*60];

		int PID = (blockIdx.y * blockDim.x + threadIdx.x) % num_particles;
		int obs_i = threadIdx.y;

		/*Step the particles*/
		Dvc_State* current_particle = NULL;

		/*make a local copy of the particle in shared memory*/
		if (obs_i == 0) {
			DvcModelCopyToShared_(
					(Dvc_State*) ((int*) localParticles + Shared_mem_per_particle * threadIdx.x),
					vnode_particles, PID % num_particles, false);
		}
		current_particle = (Dvc_State*) ((int*) localParticles + Shared_mem_per_particle * threadIdx.x);
		__syncthreads();

		float reward = 0;
		/*step the local particle, get obs and reward*/
		if(parent_action>=0)
		{
			if(DvcModelStepIntObs_)
			{
				int terminal = DvcModelStepIntObs_(*current_particle, streams->Entry(current_particle->scenario_id, streams->position_-1),
					parent_action, reward, Intobs+threadIdx.x*num_obs_elements);
			}
			else
			{
				printf("Undefined DvcModelStepIntObs_!\n");
			}
			__syncthreads();

			if (blockIdx.y * blockDim.x + threadIdx.x < num_particles) {
				/*Record stepped particles from parent as particles in this node*/
				if (obs_i == 0 &&  blockIdx.x==0) {
					Dvc_State* temp = DvcModelGet_(vnode_particles, PID % num_particles);
					DvcModelCopyNoAlloc_(temp, current_particle,0, false);
				}
			}
			__syncthreads();
		}
	}
}



__global__ void
//__launch_bounds__(64, 16)
Step_IntObs(int total_num_scenarios, int num_particles, Dvc_State* vnode_particles,
		const int* vnode_particleIDs, float* step_reward_all_a,
		int* observations_all_a_p,const int num_obs_elements,
		Dvc_State* new_particles,
		Dvc_RandomStreams* streams, bool* terminal_all_a_p,
		int Shared_mem_per_particle) {

	if (blockIdx.y * blockDim.x + threadIdx.x < num_particles) {
		__shared__ int Intobs[32*60];

		int action = blockIdx.x;
		int PID = (blockIdx.y * blockDim.x + threadIdx.x) % num_particles;
		int obs_i = threadIdx.y;

		if (blockIdx.y == 0 && threadIdx.x == 0 && obs_i == 0)
			step_reward_all_a[action] = 0;

		/*Step the particles*/
		Dvc_State* current_particle = NULL;

		/*make a local copy of the particle in shared memory*/
		if (obs_i == 0) {
			DvcModelCopyToShared_(
					(Dvc_State*) ((int*) localParticles + Shared_mem_per_particle * threadIdx.x),
					vnode_particles, PID % num_particles, false);
		}
		current_particle = (Dvc_State*) ((int*) localParticles + Shared_mem_per_particle * threadIdx.x);
		__syncthreads();

		float reward=0;
		int terminal=false;

		if(FIX_SCENARIO==1 || GPUDoPrint)
			if(GPUDoPrint && current_particle->scenario_id==PRINT_ID && blockIdx.x==ACTION_ID && threadIdx.y==0){
				printf("[GPU] step particle \n");
			}

		if(DvcModelStepIntObs_)
		{
			terminal = DvcModelStepIntObs_(*current_particle, streams->Entry(current_particle->scenario_id),
					action, reward, Intobs+threadIdx.x*num_obs_elements);
		}
		else
		{
			printf("Undefined DvcModelStepIntObs_!\n");
		}

		if(!Dvc_config->track_alpha_vector)
		{
			reward = reward * current_particle->weight;
		}
		int parent_PID = vnode_particleIDs[PID];
		/*Record stepped particles*/
		int global_list_pos = action * total_num_scenarios + parent_PID;

		if (obs_i == 0) {
			Dvc_State* temp = DvcModelGet_(new_particles, global_list_pos);
			//printf("Block id %d, thread id %d, global lis pos %d, des_i %p \n", blockIdx.x, threadIdx.x, global_list_pos, (void*) temp);
			DvcModelCopyNoAlloc_(temp, current_particle, 0, false);

			/*Record all observations for CPU usage*/
			if (!terminal) {
				for(int i=0;i<num_obs_elements;i++)
					observations_all_a_p[global_list_pos*num_obs_elements+i] = Intobs[threadIdx.x*num_obs_elements+i];
			} else {
				observations_all_a_p[global_list_pos*num_obs_elements] = 0;//no content in obs list
			}

			if(!Dvc_config->track_alpha_vector)
			{
				/*Accumulate rewards of all particles from the v-node for CPU usage*/
				atomicAdd(step_reward_all_a + action, reward);
			}
			else
			{
				step_reward_all_a[global_list_pos] = reward;
			}

			if (obs_i == 0)
				terminal_all_a_p[global_list_pos] = terminal;
		}
	}
}

/**
 * InitBounds kernel (Long observation type):
 * Calulate the uppper and lower bounds for all particles already stepped using all expansion actions
 * Y dimemsion in thread blocks are reserved for Hetergenous element-wise parallelization
 */

__global__ void
_InitBounds_LongObs(int total_num_scenarios, int num_particles,
		Dvc_State* new_particles, const int* vnode_particleIDs,
		float* upper_all_a_p, float* utility_upper_all_a_p,
		Dvc_ValuedAction* default_move_all_a_p, OBS_TYPE* observations_all_a_p,
		Dvc_RandomStreams* streams, Dvc_History* history, int depth,
		int hist_size) {
	int action = blockIdx.x;
	int PID = (blockIdx.y * blockDim.x + threadIdx.x) % num_particles;

	int parent_PID = vnode_particleIDs[PID];
	Dvc_State* current_particle = (Dvc_State*) ((int*) localParticles + 60 * threadIdx.x);

	int global_list_pos = action * total_num_scenarios + parent_PID;

	/*Copy particle from global memory to shared memory*/
	if (threadIdx.y == 0) {
		Dvc_State* temp = DvcModelGet_(new_particles, global_list_pos);
		if(DvcModelCopyToShared_)
			DvcModelCopyToShared_(current_particle, temp, 0, false);
		else
			printf("InitBound kernel: DvcModelCopyToShared_ has not been defined!\n");
	}
	__syncthreads();

	/*Do roll-out using the stepped particle*/
	Dvc_History local_history;
	local_history.currentSize_ = hist_size;
	local_history.actions_ = history->actions_;
	local_history.observations_ = history->observations_;
	if(hist_size>0)
	{
		local_history.actions_[hist_size - 1] = blockIdx.x;
		local_history.observations_[hist_size - 1] =
				observations_all_a_p[global_list_pos];
	}
	Dvc_RandomStreams local_streams(streams->num_streams_, streams->length_,
			streams->streams_,
			(hist_size>0)?streams->position_+1:streams->position_);

	float local_upper;
	if (threadIdx.y == 0 && (blockIdx.y * blockDim.x + threadIdx.x) < num_particles) {
		local_upper = DvcUpperBoundValue_(current_particle, 0, local_history);
		local_upper *= Dvc_Globals::Dvc_Discount(Dvc_config, depth);

	}

	local_streams.position(depth);
	Dvc_ValuedAction local_lower;

	if(DvcChooseEdge_)
		local_lower = DvcLowerBoundValue_( current_particle, local_streams,
				local_history, DvcChooseEdge_(action,observations_all_a_p[global_list_pos]));
	else
		local_lower = DvcLowerBoundValue_( current_particle, local_streams,
				local_history, 0);

	local_lower.value *= Dvc_Globals::Dvc_Discount(Dvc_config, depth);
	local_streams.position(depth);

	/*Prepare data for returning to host*/
	if (threadIdx.y == 0 && (blockIdx.y * blockDim.x + threadIdx.x) < num_particles) {
		global_list_pos=action * total_num_scenarios + PID;
		if(!Dvc_config->track_alpha_vector)
		{
			local_lower.value = local_lower.value * current_particle->weight;
			local_upper = local_upper * current_particle->weight;
		}
		utility_upper_all_a_p[global_list_pos] = local_upper;

		upper_all_a_p[global_list_pos] = local_upper;
		default_move_all_a_p[global_list_pos] = local_lower;
	}
}

/**
 * InitBounds kernel (Int array observation type):
 * Calulate the uppper and lower bounds for all particles already stepped using all expansion actions
 * Y dimemsion in thread blocks are reserved for Hetergenous element-wise parallelization
 */

__global__ void
_InitBounds_IntArrayObs(int total_num_scenarios, int num_particles,
		Dvc_State* new_particles, const int* vnode_particleIDs,
		float* upper_all_a_p, float* utility_upper_all_a_p,
		Dvc_ValuedAction* default_move_all_a_p, OBS_TYPE* observations_all_a_p,
		Dvc_RandomStreams* streams, Dvc_History* history, int depth,
		int hist_size,int Shared_mem_per_particle) {

	int action = blockIdx.x;

	if (blockIdx.y * blockDim.x + threadIdx.x < num_particles) {
		int PID = (blockIdx.y * blockDim.x + threadIdx.x) % num_particles;
		Dvc_State* current_particle = NULL;

		int parent_PID = vnode_particleIDs[PID];
		current_particle = (Dvc_State*) ((int*) localParticles + Shared_mem_per_particle * threadIdx.x);

		int global_list_pos = action * total_num_scenarios + parent_PID;

		/*Copy particle from global memory to shared memory*/
		if (threadIdx.y == 0) {
			Dvc_State* temp = DvcModelGet_(new_particles, global_list_pos);
			if(DvcModelCopyToShared_)
				DvcModelCopyToShared_(current_particle, temp, 0, false);
			else
				printf("InitBound kernel: DvcModelCopyToShared_ has not been defined!\n");
		}
		__syncthreads();

		//Do roll-out using the updated particle
		Dvc_History local_history;
		local_history.currentSize_ = hist_size;
		local_history.actions_ = history->actions_;
		local_history.observations_ = history->observations_;

		Dvc_RandomStreams local_streams(streams->num_streams_, streams->length_,
				streams->streams_,
				(hist_size>0)?streams->position_+1:streams->position_);


		float local_upper;
		if (threadIdx.y == 0 && (blockIdx.y * blockDim.x + threadIdx.x) < num_particles) {

			local_upper = DvcUpperBoundValue_(current_particle, 0, local_history);
			local_upper *= Dvc_Globals::Dvc_Discount(Dvc_config, depth);
		}


		//Lower bound
		local_streams.position(depth);
		Dvc_ValuedAction local_lower;
		if(DvcChooseEdge_)
			local_lower = DvcLowerBoundValue_( current_particle, local_streams,
					local_history, DvcChooseEdge_(action,observations_all_a_p[global_list_pos]));
		else
			local_lower = DvcLowerBoundValue_( current_particle, local_streams,
					local_history, 0);

		local_lower.value *= Dvc_Globals::Dvc_Discount(Dvc_config, depth);
		local_streams.position(depth);


		/*Prepare data for returning to host*/
		if (threadIdx.y == 0 && (blockIdx.y * blockDim.x + threadIdx.x) < num_particles) {
			global_list_pos=action * total_num_scenarios + PID;
			if(!Dvc_config->track_alpha_vector)
			{
				local_lower.value = local_lower.value * current_particle->weight;
				local_upper = local_upper * current_particle->weight;
			}
			utility_upper_all_a_p[global_list_pos] = local_upper;

			upper_all_a_p[global_list_pos] = local_upper;
			default_move_all_a_p[global_list_pos] = local_lower;
		}
	}
}

__global__ void
_CalObsProb_LongObs(int total_num_scenarios, int num_particles,
		Dvc_State* new_particles, const int* vnode_particleIDs, bool* term_all_a_p,
		float* obs_prob_all_a_p_obs, OBS_TYPE* observations_all_a_p)
{
	int s_mult_o = num_particles*num_particles;
	int action = blockIdx.x;
		int s_mult_o_id = (blockIdx.y * blockDim.x + threadIdx.x) % (s_mult_o);
		int obs_id = vnode_particleIDs[s_mult_o_id/num_particles];
		int scenario_id = vnode_particleIDs[s_mult_o_id % num_particles];
		//int parent_PID = vnode_particleIDs[PID];
		Dvc_State* current_particle = (Dvc_State*) ((int*) localParticles + 60 * threadIdx.x);

		int particle_list_pos = action * total_num_scenarios + scenario_id;
		int obs_list_pos = action * total_num_scenarios + obs_id;

		/*Copy particle from global memory to shared memory*/
		if (threadIdx.y == 0) {
			Dvc_State* temp = DvcModelGet_(new_particles, particle_list_pos);
			if(DvcModelCopyToShared_)
				DvcModelCopyToShared_(current_particle, temp, 0, false);
			else
				printf("CalObsProb kernel: DvcModelCopyToShared_ has not been defined!\n");
		}
		__syncthreads();

		/*Calculate obs prob for stepped particle*/




		float obs_prob;
		if (threadIdx.y == 0 && (blockIdx.y * blockDim.x + threadIdx.x) < s_mult_o) {
			if(term_all_a_p[particle_list_pos] == false  && term_all_a_p[obs_list_pos] == false)
			{
			obs_prob = DvcModelObsProb_(observations_all_a_p[obs_list_pos], *current_particle, action);


		/*Prepare data for returning to host*/

			int global_list_pos=(action * total_num_scenarios*total_num_scenarios) + (obs_list_pos*total_num_scenarios) + scenario_id;
			obs_prob_all_a_p_obs[global_list_pos] = obs_prob;
			}
		}
}
__global__ void
_CalObsProb_IntArrayObs(int total_num_scenarios, int num_particles,
		Dvc_State* new_particles, const int* vnode_particleIDs, bool* term_all_a_p,
		float* obs_prob_all_a_p_obs, int* observations_all_a_p,const int num_obs_elements,
		int Shared_mem_per_particle)
{
	int s_mult_o = num_particles*num_particles;
		int action = blockIdx.x;
			int s_mult_o_id = (blockIdx.y * blockDim.x + threadIdx.x) % (s_mult_o);
			int obs_id = vnode_particleIDs[s_mult_o_id/num_particles];
			int scenario_id = vnode_particleIDs[s_mult_o_id % num_particles];
			//int parent_PID = vnode_particleIDs[PID];
			Dvc_State* current_particle = (Dvc_State*) ((int*) localParticles + Shared_mem_per_particle* threadIdx.x);

			int particle_list_pos = action * total_num_scenarios + scenario_id;
			int obs_list_pos = action * total_num_scenarios + obs_id;

			/*Copy particle from global memory to shared memory*/
			if (threadIdx.y == 0) {
				Dvc_State* temp = DvcModelGet_(new_particles, particle_list_pos);
				if(DvcModelCopyToShared_)
					DvcModelCopyToShared_(current_particle, temp, 0, false);
				else
					printf("CalObsProb kernel: DvcModelCopyToShared_ has not been defined!\n");
			}
			__syncthreads();

			/*Calculate obs prob for stepped particle*/

			float obs_prob;
			if (threadIdx.y == 0 && (blockIdx.y * blockDim.x + threadIdx.x) < s_mult_o) {
				if(term_all_a_p[particle_list_pos] == false  && term_all_a_p[obs_list_pos] == false)
				{
				int Intobs[200]; //not compiling with num_obs_elements
				for(int i=0;i<num_obs_elements;i++)
				{
					Intobs[i] = observations_all_a_p[obs_list_pos*num_obs_elements+i] ;
				}
				obs_prob = DvcModelObsProbIntObs_(Intobs, *current_particle, action);


			/*Prepare data for returning to host*/

				int global_list_pos=(action * total_num_scenarios*total_num_scenarios) + (obs_id*total_num_scenarios) + scenario_id;
				obs_prob_all_a_p_obs[global_list_pos] = obs_prob;
				}
			}

}


/**
 * PrepareGPUDataForNode function:
 * Calulate the uppper and lower bounds for all particles already stepped using all expansion actions
 * Y dimemsion in thread blocks are reserved for Hetergenous element-wise parallelization
 */
void DESPOT::PrepareGPUDataForNode(VNode* vnode,const DSPOMDP* model, int ThreadID,RandomStreams& streams)
{
#ifdef RECORD_TIME
	auto start = Time::now();
#endif
	streams.position(vnode->depth());
	
		const std::vector<State*>& particles = vnode->particles();
		const std::vector<int>& particleIDs = vnode->particleIDs();
		int NumParticles = particleIDs.size();

		/*Copy particle IDs in the new node to the ID list in device memory*/
		/*Needed with alpha vector update to not step terminal particles*/


		model->CopyParticleIDsToGPU(Dvc_particleIDs_long[ThreadID], particleIDs,
			&Globals::GetThreadCUDAStream(ThreadID));
		if(!Globals::config.track_alpha_vector)

		{
		if(vnode->parent()!=NULL) // New node but not root node
		{
			/*Create GPU particles for the new v-node*/
			Dvc_State* new_particles = model->AllocGPUParticles(
					NumParticles, MEMORY_MODE(ALLOC));

			/*Copy parent particles to the new particle list*/
			model->CopyGPUParticlesFromParent(new_particles,
					vnode->parent()->parent()->GetGPUparticles(), // parent vnode particles
					0, Dvc_particleIDs_long[ThreadID],
					NumParticles,true,
					Dvc_streams[ThreadID], streams.position(),
					&Globals::GetThreadCUDAStream(ThreadID));

			/*Link the new particle list to the new node*/
			vnode->AssignGPUparticles(new_particles,
					NumParticles);

			vnode->weight_=NumParticles/((float)Globals::config.num_scenarios);
		}
	}

#ifdef RECORD_TIME

	double oldValue=CopyParticleTime.load();
	CopyParticleTime.compare_exchange_weak(oldValue,oldValue+ Globals::ElapsedTime(start));
#endif
}

void DESPOT::PrepareGPUDataForCommonQNode(QNode* qnode, const DSPOMDP* model, int ThreadID, RandomStreams& streams, const std::vector<int>& particleIDs )
{
#ifdef RECORD_TIME
	auto start = Time::now();
#endif
	//streams.position(vnode->depth());
	if(Globals::config.track_alpha_vector)

	{
		int NumParticles = particleIDs.size();

		/*Copy particle IDs in the new node to the ID list in device memory*/
		/*Not needed with alpha vector update as expanded particles are not thrown*/


		model->CopyParticleIDsToGPU(Dvc_particleIDs_long[ThreadID], particleIDs,
			&Globals::GetThreadCUDAStream(ThreadID));


			/*Create GPU particles for the new v-node*/
			Dvc_State* new_particles = model->AllocGPUParticles(
					NumParticles, MEMORY_MODE(ALLOC));

			/*Copy parent particles to the new particle list*/
			model->CopyGPUParticlesFromParent(new_particles,
					Dvc_stepped_particles_all_a[ThreadID], // parent vnode particles
					0, Dvc_particleIDs_long[ThreadID],
					NumParticles,true,
					Dvc_streams[ThreadID], streams.position(),
					&Globals::GetThreadCUDAStream(ThreadID));

			/*Link the new particle list to the new node*/
			int particle_offset = 0;
			for(int action = 0; action < qnode->common_children_.size(); action++)
			{
				logd << "Action " << action << " Particle offset " << particle_offset << std::endl;
				qnode->common_children_[action]->GPU_particles_ = model->GetPointerToParticleList(particle_offset, new_particles);
				//qnode->common_children_[action]->num_GPU_particles_ = qnode->common_children_[action]->particleIDs_.size();
				particle_offset = particle_offset + qnode->common_children_[action]->num_GPU_particles_;
				/*int action = particleIDs[i]/Globals::config.num_scenarios;
				int scenario_id = particleIDs[i] % Globals::config.num_scenarios;
				assert(new_particles[i].scenario_id == scenario_id);
				if(qnode->common_children_[action]->GPU_particles_ == NULL)
				{
					qnode->common_children_[action]->GPU_particles_ = new_particles + i;
					qnode->common_children_[action]->num_GPU_particles_ = 1;
				}
				else
				{
					qnode->common_children_[action]->num_GPU_particles_ = qnode->common_children_[action]->num_GPU_particles_ + 1;
				}*/
			}

	}

#ifdef RECORD_TIME

	double oldValue=CopyParticleTime.load();
	CopyParticleTime.compare_exchange_weak(oldValue,oldValue+ Globals::ElapsedTime(start));
#endif
}
void DESPOT::MCSimulation(VNode* vnode, int ThreadID,
		const DSPOMDP* model, RandomStreams& streams,History& history, bool Do_rollout)
{
	if((FIX_SCENARIO==1 || DESPOT::Print_nodes) && vnode->parent()==NULL){
		GPUDoPrint=true;
	}
#ifdef RECORD_TIME
	auto start = Time::now();
#endif

	int blocky;
	dim3 GridDim;
	dim3 ThreadDim;
	int NumActions = model->NumActions();
	int NumObs = model->NumObservations();
	int NumParticles=vnode->num_GPU_particles_;
	int ParalllelisminStep = model->ParallelismInStep();
	int Shared_mem_per_particle=CalSharedMemSize();

	int threadx = 32;
	blocky =
			(NumParticles % threadx == 0) ?
					NumParticles / threadx : NumParticles / threadx + 1;
	GridDim.x = NumActions;
	GridDim.y = blocky;
	ThreadDim.x = threadx;
	ThreadDim.y = model->ParallelismInStep();

	logd << "[DESPOT::MCSimulation] Step GPU particles "<< endl;


	logd << "vnode->GetGPUparticles() = "<< vnode->GetGPUparticles() ;
	if(Globals::config.track_alpha_vector)
	{
		logd << "Dvc_r_all_a_and_p = "<< Dvc_r_all_a_and_p ;
	}
	else
	{
		 logd << "Dvc_r_all_a = "<< Dvc_r_all_a ;
	}
		 logd << " Dvc_obs_int_all_a_and_p = "<< Dvc_obs_int_all_a_and_p<<
		 " Dvc_stepped_particles_all_a = " << Dvc_stepped_particles_all_a <<
		 " Dvc_streams = " << Dvc_streams <<
		 " Dvc_term_all_a_and_p = " << Dvc_term_all_a_and_p <<
		 " Dvc_particleIDs_long = " << Dvc_particleIDs_long <<
		 endl;

	logd << "vnode->GetGPUparticles() = "<< vnode->GetGPUparticles();
	if(Globals::config.track_alpha_vector)
		{
			logd << "Dvc_r_all_a_and_p[ThreadID] = "<< Dvc_r_all_a_and_p[ThreadID] ;
		}
		else
		{
		 logd << " Dvc_r_all_a[ThreadID] = "<< Dvc_r_all_a[ThreadID] ;
		}
	logd << " Dvc_obs_int_all_a_and_p[ThreadID] = "<< Dvc_obs_int_all_a_and_p[ThreadID]<<
		 " Dvc_stepped_particles_all_a[ThreadID] = " << Dvc_stepped_particles_all_a[ThreadID] <<
		 " Dvc_streams[ThreadID] = " << Dvc_streams[ThreadID] <<
		 " Dvc_term_all_a_and_p[ThreadID] = " << Dvc_term_all_a_and_p[ThreadID] <<
		 " Dvc_particleIDs_long[ThreadID] = " << Dvc_particleIDs_long[ThreadID] <<
		 endl;


	if(Obs_type==OBS_INT_ARRAY)
	{
		if(GPUDoPrint || DESPOT::Print_nodes){
			printf("pre-step particle %d\n", vnode->GetGPUparticles());
			printf("do rollout = %d\n", Do_rollout);
		}
		int num_Obs_element=num_Obs_element_in_GPU;
		if (Globals::config.use_multi_thread_){
			if(!Globals::config.track_alpha_vector){
				PreStep_IntObs<<<dim3(1, GridDim.y), ThreadDim, threadx * Shared_mem_per_particle * sizeof(int),
					Globals::GetThreadCUDAStream(ThreadID)>>>(
					NumParticles,
					vnode->GetGPUparticles(),
					num_Obs_element,
					Dvc_streams[ThreadID],
					(vnode->parent()==NULL)?-1:vnode->parent()->edge(),
					Shared_mem_per_particle);

			if (Do_rollout)
				Step_IntObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int),
						Globals::GetThreadCUDAStream(ThreadID)>>>(Globals::config.num_scenarios,
						NumParticles,
						vnode->GetGPUparticles(),
						Dvc_particleIDs_long[ThreadID], Dvc_r_all_a[ThreadID],
						Dvc_obs_int_all_a_and_p[ThreadID],num_Obs_element,
						Dvc_stepped_particles_all_a[ThreadID],
						Dvc_streams[ThreadID],
						Dvc_term_all_a_and_p[ThreadID],
						Shared_mem_per_particle);
			}
			else
			{

				if (Do_rollout)
								Step_IntObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int),
										Globals::GetThreadCUDAStream(ThreadID)>>>(Globals::config.num_scenarios,
										NumParticles,
										vnode->GetGPUparticles(),
										Dvc_particleIDs_long[ThreadID], Dvc_r_all_a_and_p[ThreadID],
										Dvc_obs_int_all_a_and_p[ThreadID],num_Obs_element,
										Dvc_stepped_particles_all_a[ThreadID],
										Dvc_streams[ThreadID],
										Dvc_term_all_a_and_p[ThreadID],
										Shared_mem_per_particle);

			}
		}
		else{
			if(!Globals::config.track_alpha_vector){
				PreStep_IntObs<<<dim3(1, GridDim.y), ThreadDim, threadx * Shared_mem_per_particle * sizeof(int)>>>
					(NumParticles,
					vnode->GetGPUparticles(),
					num_Obs_element,
					Dvc_streams[ThreadID],
					(vnode->parent()==NULL)?-1:vnode->parent()->edge(),
					Shared_mem_per_particle);

			if (Do_rollout)
				Step_IntObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int)>>>
					(Globals::config.num_scenarios,
					NumParticles,
					vnode->GetGPUparticles(),
					Dvc_particleIDs_long[ThreadID], Dvc_r_all_a[ThreadID],
					Dvc_obs_int_all_a_and_p[ThreadID],num_Obs_element,
					Dvc_stepped_particles_all_a[ThreadID],
					Dvc_streams[ThreadID],
					Dvc_term_all_a_and_p[ThreadID],
					Shared_mem_per_particle);
			}
			else
			{

				if (Do_rollout)
				{
								//DESPOT::CheckSteppedParticles(NumActions);
								//std::cout << "Thread id is " << ThreadID << std::endl;
								Step_IntObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int)>>>
									(Globals::config.num_scenarios,
									NumParticles,
									vnode->GetGPUparticles(),
									Dvc_particleIDs_long[ThreadID], Dvc_r_all_a_and_p[ThreadID],
									Dvc_obs_int_all_a_and_p[ThreadID],num_Obs_element,
									Dvc_stepped_particles_all_a[ThreadID],
									Dvc_streams[ThreadID],
									Dvc_term_all_a_and_p[ThreadID],
									Shared_mem_per_particle);
									}
			}
		}
	}
	else
	{
		if (Globals::config.use_multi_thread_){
			if(!Globals::config.track_alpha_vector){
			PreStep_LongObs<<<dim3(1, GridDim.y), ThreadDim, threadx * Shared_mem_per_particle * sizeof(int),
					Globals::GetThreadCUDAStream(ThreadID)>>>(Globals::config.num_scenarios,
					NumParticles, vnode->GetGPUparticles(),
					Dvc_particleIDs_long[ThreadID], Dvc_streams[ThreadID],
					(vnode->parent()==NULL)?-1:vnode->parent()->edge());

			if (Do_rollout)
				Step_LongObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int),
						Globals::GetThreadCUDAStream(ThreadID)>>>(Globals::config.num_scenarios,
						NumParticles, vnode->GetGPUparticles(),
						Dvc_particleIDs_long[ThreadID], Dvc_r_all_a[ThreadID],
						Dvc_obs_all_a_and_p[ThreadID],
						Dvc_stepped_particles_all_a[ThreadID],
						Dvc_streams[ThreadID],
						Dvc_term_all_a_and_p[ThreadID]);
			}
			else
			{
				if (Do_rollout)
								Step_LongObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int),
										Globals::GetThreadCUDAStream(ThreadID)>>>(Globals::config.num_scenarios,
										NumParticles, vnode->GetGPUparticles(),
										Dvc_particleIDs_long[ThreadID], Dvc_r_all_a_and_p[ThreadID],
										Dvc_obs_all_a_and_p[ThreadID],
										Dvc_stepped_particles_all_a[ThreadID],
										Dvc_streams[ThreadID],
										Dvc_term_all_a_and_p[ThreadID]);
			}
		}
		else{
			if(!Globals::config.track_alpha_vector)
			{
			PreStep_LongObs<<<dim3(1, GridDim.y), ThreadDim, threadx * Shared_mem_per_particle * sizeof(int)>>>
					(Globals::config.num_scenarios,
					NumParticles, vnode->GetGPUparticles(),
					Dvc_particleIDs_long[ThreadID], Dvc_streams[ThreadID],
					(vnode->parent()==NULL)?-1:vnode->parent()->edge());

			if (Do_rollout)
				Step_LongObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int)>>>
						(Globals::config.num_scenarios,
						NumParticles, vnode->GetGPUparticles(),
						Dvc_particleIDs_long[ThreadID], Dvc_r_all_a[ThreadID],
						Dvc_obs_all_a_and_p[ThreadID],
						Dvc_stepped_particles_all_a[ThreadID],
						Dvc_streams[ThreadID],
						Dvc_term_all_a_and_p[ThreadID]);
			}
			else
			{
				if (Do_rollout)
								Step_LongObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int)>>>
										(Globals::config.num_scenarios,
										NumParticles, vnode->GetGPUparticles(),
										Dvc_particleIDs_long[ThreadID], Dvc_r_all_a_and_p[ThreadID],
										Dvc_obs_all_a_and_p[ThreadID],
										Dvc_stepped_particles_all_a[ThreadID],
										Dvc_streams[ThreadID],
										Dvc_term_all_a_and_p[ThreadID]);
			}
		}
	}

#ifdef RECORD_TIME
	double oldValue=AveRewardTime.load();
	AveRewardTime.compare_exchange_weak(oldValue,oldValue+
					chrono::duration_cast < ns
					> (Time::now() - start).count()/1000000000.0f);
#endif

	logd << "[DESPOT::MCSimulation] Rollout GPU particles "<< endl;

	if(Do_rollout)
	{

	#ifdef RECORD_TIME
		start = Time::now();
	#endif

		if(Obs_type==OBS_INT_ARRAY)
		{
			if(GPUDoPrint || DESPOT::Print_nodes){
				printf("rollout particle %d\n", vnode->GetGPUparticles() );
			}
			if (Globals::config.use_multi_thread_)
				_InitBounds_IntArrayObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int),
						Globals::GetThreadCUDAStream(ThreadID)>>>(Globals::config.num_scenarios,
						NumParticles, Dvc_stepped_particles_all_a[ThreadID],
						Dvc_particleIDs_long[ThreadID], Dvc_ub_all_a_p[ThreadID],
						Dvc_uub_all_a_p[ThreadID], Dvc_lb_all_a_p[ThreadID],
						Dvc_obs_all_a_and_p[ThreadID], Dvc_streams[ThreadID],
						Dvc_history[ThreadID], vnode->depth() + 1,
						history.Size() + 1,Shared_mem_per_particle);
			else
				_InitBounds_IntArrayObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int)>>>(
						Globals::config.num_scenarios, NumParticles,
						Dvc_stepped_particles_all_a[ThreadID],
						Dvc_particleIDs_long[ThreadID], Dvc_ub_all_a_p[ThreadID],
						Dvc_uub_all_a_p[ThreadID], Dvc_lb_all_a_p[ThreadID],
						Dvc_obs_all_a_and_p[ThreadID],Dvc_streams[ThreadID],
						Dvc_history[ThreadID], vnode->depth() + 1,
						history.Size() + 1,Shared_mem_per_particle);
		}
		else
		{
			if (Globals::config.use_multi_thread_)
				_InitBounds_LongObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int),
						Globals::GetThreadCUDAStream(ThreadID)>>>(Globals::config.num_scenarios,
						NumParticles, Dvc_stepped_particles_all_a[ThreadID],
						Dvc_particleIDs_long[ThreadID], Dvc_ub_all_a_p[ThreadID],
						Dvc_uub_all_a_p[ThreadID], Dvc_lb_all_a_p[ThreadID],
						Dvc_obs_all_a_and_p[ThreadID], Dvc_streams[ThreadID],
						Dvc_history[ThreadID], vnode->depth() + 1,
						history.Size() + 1);
			else
				_InitBounds_LongObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int)>>>(
						Globals::config.num_scenarios, NumParticles,
						Dvc_stepped_particles_all_a[ThreadID],
						Dvc_particleIDs_long[ThreadID], Dvc_ub_all_a_p[ThreadID],
						Dvc_uub_all_a_p[ThreadID], Dvc_lb_all_a_p[ThreadID],
						Dvc_obs_all_a_and_p[ThreadID], Dvc_streams[ThreadID],
						Dvc_history[ThreadID], vnode->depth() + 1,
						history.Size() + 1);
		}


	logd << "[DESPOT::MCSimulation] Read back GPU data "<< endl;

	#ifdef RECORD_TIME
		oldValue=InitBoundTime.load();
		InitBoundTime.compare_exchange_weak(oldValue,oldValue + Globals::ElapsedTime(start));
		start = Time::now();
	#endif
		if(Globals::config.track_alpha_vector)
		{

			if(Do_rollout)
			{
				int s_mult_o = NumParticles*NumParticles;
				blocky = (s_mult_o % threadx == 0) ?
									s_mult_o / threadx : s_mult_o / threadx + 1;
					GridDim.x = NumActions;
					GridDim.y = blocky;
					ThreadDim.x = threadx;
					ThreadDim.y = model->ParallelismInStep();
				if(Obs_type==OBS_INT_ARRAY)
				{
					int num_Obs_element=num_Obs_element_in_GPU;
					if(GPUDoPrint || DESPOT::Print_nodes){
						printf("obs prob particle %d\n", vnode->GetGPUparticles() );
					}
					if (Globals::config.use_multi_thread_)
						_CalObsProb_IntArrayObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int),
								Globals::GetThreadCUDAStream(ThreadID)>>>(Globals::config.num_scenarios,
								NumParticles, Dvc_stepped_particles_all_a[ThreadID],
								Dvc_particleIDs_long[ThreadID], Dvc_term_all_a_and_p[ThreadID], Dvc_obs_prob_all_a_p_obs[ThreadID],
								Dvc_obs_int_all_a_and_p[ThreadID], num_Obs_element, Shared_mem_per_particle);
					else
						_CalObsProb_IntArrayObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int)>>>(
								Globals::config.num_scenarios, NumParticles,
								Dvc_stepped_particles_all_a[ThreadID],
								Dvc_particleIDs_long[ThreadID], Dvc_term_all_a_and_p[ThreadID], Dvc_obs_prob_all_a_p_obs[ThreadID],
								Dvc_obs_int_all_a_and_p[ThreadID] ,num_Obs_element,Shared_mem_per_particle);
				}
				else
				{
					if (Globals::config.use_multi_thread_)
						_CalObsProb_LongObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int),
								Globals::GetThreadCUDAStream(ThreadID)>>>(Globals::config.num_scenarios,
								NumParticles, Dvc_stepped_particles_all_a[ThreadID],
								Dvc_particleIDs_long[ThreadID], Dvc_term_all_a_and_p[ThreadID], Dvc_obs_prob_all_a_p_obs[ThreadID],
								Dvc_obs_all_a_and_p[ThreadID]);
					else
						_CalObsProb_LongObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int)>>>(
								Globals::config.num_scenarios, NumParticles,
								Dvc_stepped_particles_all_a[ThreadID],
								Dvc_particleIDs_long[ThreadID], Dvc_term_all_a_and_p[ThreadID], Dvc_obs_prob_all_a_p_obs[ThreadID],
								Dvc_obs_all_a_and_p[ThreadID]);
				}
			}
		}




		ReadBackData(ThreadID);

	#ifdef RECORD_TIME
		oldValue=DataBackTime.load();
		DataBackTime.compare_exchange_weak(oldValue,oldValue+ Globals::ElapsedTime(start));
	#endif
	}

	if((FIX_SCENARIO==1 || DESPOT::Print_nodes) && vnode->parent()==NULL)
	{
		HANDLE_ERROR(hipDeviceSynchronize());
		GPUDoPrint=false;
	}
}

void DESPOT::GPU_Expand_Action(VNode* vnode, ScenarioLowerBound* lb,
		ScenarioUpperBound* ub, const DSPOMDP* model, RandomStreams& streams,
		History& history) {

	int ThreadID = 0;
	if (Globals::config.use_multi_thread_)
		ThreadID = Globals::MapThread(this_thread::get_id());
	int NumActions = model->NumActions();
	int NumObs = model->NumObservations();
	int NumScenarios = Globals::config.num_scenarios;

	Globals::Global_print_expand(this_thread::get_id(), vnode, vnode->depth(), vnode->edge());

	if(Globals::config.use_multi_thread_)
		static_cast<Shared_VNode*>(vnode)->is_waiting_=true;

	HitCount++;
	Globals::AddExpanded();
	auto start_total = Time::now();

	/*Update streams, history, and particles into GPU*/
	PrepareGPUDataForNode(vnode, model, ThreadID, streams);


	int NumParticles = vnode->particleIDs().size();
	/*std::cout << "Num particles : " << NumParticles << "," << vnode->num_GPU_particles_ << std::endl;
	std::cout << "Particles ids " ;
	for(int i = 0; i < vnode->particleIDs().size(); i++)
	{
		std::cout << vnode->particleIDs()[i] << " ";
	}
	std::cout << std::endl;
	*/
	AveNumParticles = AveNumParticles * (HitCount - 1) / HitCount
			+ NumActions * NumParticles / HitCount;

	
	/*Run Monte Carlo simulations in GPU: update particles and perform rollouts*/
	MCSimulation(vnode, ThreadID,model, streams,history,true);


	/*Debug lb*/
	if(false)
	{
		for(int i=0;i<NumParticles;i++)
		{
			cout.precision(3);
			cout<<Hst_lb_all_a_p[ThreadID][i].action<<"/"<<Hst_lb_all_a_p[ThreadID][i].value<<" ";
			cout<<Hst_ub_all_a_p[ThreadID][i]<<" ";
		}
		cout<<endl;
	}
	/*Debug lb*/
	std::vector<int> particleIDs=vnode->particleIDs();
	if(Globals::config.track_alpha_vector)
	{
		/*Expand common QNode*/
		std::vector<int> particleIds_all_a;
		logd << "Num particles " << NumParticles << "[";
		/*for (int i = 0; i < NumParticles; i++) {
		    logd << particleIDs[i] << "," ;
		}*/
		logd << std::endl;
		for (int action = 0; action < NumActions; action++) {
			#ifdef RECORD_TIME
				auto start = Time::now();
			#endif
			logd << "GPU Expand Action " << action << std::endl;
			QNode* qnode;
			if(action >= vnode->children().size())
			{
				//Create new Qnode for action
				if (Globals::config.use_multi_thread_)
					qnode = new Shared_QNode(static_cast<Shared_VNode*>(vnode), action);
				else
					qnode = new QNode(vnode, action);

				vnode->children().push_back(qnode);
			}
			else
			{
			qnode = vnode->Child(action);
			}
			if(Globals::config.use_multi_thread_ && Globals::config.exploration_mode==UCT)
						static_cast<Shared_QNode*>(qnode)->visit_count_=1.1;
			vnode->common_parent_->common_children_.push_back(qnode);
			QNode* common_qnode = qnode;
			common_qnode->populating_node = qnode;
			common_qnode->step_reward_vector.resize(Globals::config.num_scenarios,0);
			if(Globals::config.use_sawtooth_upper_bound)
			{
				common_qnode->vnode_upper_bound_per_particle.resize(Globals::config.num_scenarios, 0);
			}
			double step_reward = 0;
			std::map<OBS_TYPE, std::vector<int> > partitions;
			std::map<OBS_TYPE, VNode*>& children = qnode->children();
			for (int i = 0; i < NumParticles; i++) {
			    
				int parent_PID = particleIDs[i]; // parent_PID corresponds to scenario id
				logd << "Steppig particle " << parent_PID << std::endl;
				common_qnode->step_reward_vector[parent_PID] = Globals::Discount(vnode->depth()) * Hst_r_all_a_and_p[ThreadID][action * NumScenarios
									+ parent_PID];
				step_reward += Hst_r_all_a_and_p[ThreadID][action * NumScenarios
				       									+ parent_PID] * vnode->particle_weights[parent_PID];
				OBS_TYPE obs;

				if(Obs_type==OBS_INT_ARRAY)
				{
					std::vector<int> tempobs;
					int* Int_obs_list = &Hst_obs_int_all_a_and_p[ThreadID][(action * NumScenarios
										+ parent_PID)*num_Obs_element_in_GPU];
					int num_obs_elements=Int_obs_list[0];
					tempobs.resize(num_obs_elements);

					logd << "Received obs ";
					for(int j=0;j<num_obs_elements;j++)
					{
						tempobs[j]=Int_obs_list[j+1];
						logd << tempobs[j] << " ";
					}

					std::hash<std::vector<int>> myhash;
					obs=myhash(tempobs);
					logd << "Obs hash " << obs << std::endl;
					
				}
				else
				{
					obs = Hst_obs_all_a_and_p[ThreadID][action * NumScenarios
						+ parent_PID];
				}




				//False flag for debugging
				/*if(parent_PID == 2)
				{
					Hst_term_all_a_and_p[ThreadID][action * NumScenarios + parent_PID] = true;
				}
				*/
				if (Hst_term_all_a_and_p[ThreadID][action * NumScenarios + parent_PID] == false ) {

					particleIds_all_a.push_back(action * NumScenarios
								+ parent_PID);
					common_qnode->particleIDs_.push_back(parent_PID);
					common_qnode->particles_.push_back(NULL); //Required to keep size of particles_ consistent with particleIds
					partitions[obs].push_back(parent_PID);
				}



			} //Loop over NumParticles
#ifdef RECORD_TIME
		double oldValue=MakePartitionTime.load();
		MakePartitionTime.compare_exchange_weak(oldValue,oldValue+ Globals::ElapsedTime(start));
		/*Create new v-nodes for partitions, calculate the bounds*/
		auto nodestart = Time::now();
#endif
			step_reward = Globals::Discount(vnode->depth()) * step_reward
					- Globals::config.pruning_constant;//pruning_constant is used for regularization

			qnode->step_reward = step_reward;
			common_qnode->num_GPU_particles_ = common_qnode->particleIDs_.size();
			VNode* residual_vnode;
			logd << "Particles survived " << common_qnode->particleIDs_.size()  << std::endl;

			if(common_qnode->particleIDs_.size() > 0)
			{
				if (Globals::config.use_multi_thread_)
						{
							residual_vnode = new Shared_VNode(vnode->depth() + 1,
													 static_cast<Shared_QNode*>(qnode), static_cast<Shared_QNode*>(common_qnode),
													 Globals::RESIDUAL_OBS);
							if (Globals::config.exploration_mode == UCT)
								static_cast<Shared_VNode*>(residual_vnode)->visit_count_ = 1.1;
						}
						else
						{
							residual_vnode = new VNode(vnode->depth() + 1,
									qnode,common_qnode, Globals::RESIDUAL_OBS);
						}
					children[Globals::RESIDUAL_OBS] = residual_vnode;
				//residual_vnode->observation_particle_size = 1; //Not used anywhere probably
				residual_vnode->extra_node = true;
				residual_vnode->obs_probs_holder = residual_vnode;
				residual_vnode->obs_probs.resize(Globals::config.num_scenarios, 0);
				residual_vnode->num_GPU_particles_ = common_qnode->particleIDs_.size();

			}

			//Create child nodes
	        double max_prob_sum = 0.0;
		for (std::map<OBS_TYPE, std::vector<int> >::iterator it = partitions.begin();
			it != partitions.end(); it++) {
			OBS_TYPE obs = it->first;
	                //int observation_particle_size_ = partitions[obs].size();
			VNode* child_vnode;
			if (Globals::config.use_multi_thread_)
			        			{
			        				child_vnode = new Shared_VNode(vnode->depth() + 1,
			        				                         static_cast<Shared_QNode*>(qnode), static_cast<Shared_QNode*>(common_qnode),
			        				                         obs);
			        				if (Globals::config.exploration_mode == UCT)
			        					static_cast<Shared_VNode*>(child_vnode)->visit_count_ = 1.1;
			        			}
			        			else
			        			{
			        				child_vnode = new VNode(vnode->depth() + 1,
			        							qnode, common_qnode, obs);
			        			}
	                //vnode->observation_particle_size = observation_particle_size_;
	                child_vnode->obs_probs.resize(Globals::config.num_scenarios, 0);
			logd << " New node created!" << std::endl;
			children[obs] = child_vnode;
	                child_vnode->obs_probs_holder = child_vnode;
	                child_vnode->num_GPU_particles_ = common_qnode->particleIDs_.size();
	                if(obs == Globals::RESIDUAL_OBS)
	                {
	                    child_vnode->extra_node = true;
	                }
			double total_weight = 0;
	                for(int i = 0; i < common_qnode->particleIDs_.size();i++)
	                {
	                	int scenario_id = common_qnode->particleIDs_[i];
	                    double prob = Hst_obs_prob_all_a_p_obs[ThreadID][action * NumScenarios*NumScenarios + (it->second[0]*NumScenarios) + scenario_id];
	                    //int scenario_id = common_qnode->particles_[i]->scenario_id;
	                    //prob = model->ObsProb(obs, *common_qnode->particles_[i], qnode->edge());


	                   logd << "Obs Prob: for obs " <<  obs << " " << prob << " ";

			 // Terminal state is not required to be explicitly represented and may not have any observation
				child_vnode->particle_weights[common_qnode->particleIDs_[i]] = vnode->particle_weights[common_qnode->particleIDs_[i]]* prob;
				total_weight += child_vnode->particle_weights[common_qnode->particleIDs_[i]];
	                        //Total weight should not be zero as one particle actually produced that observation
	                        child_vnode->obs_probs[common_qnode->particleIDs_[i]] = prob;

	                        residual_vnode->obs_probs[common_qnode->particleIDs_[i]] =  residual_vnode->obs_probs[common_qnode->particleIDs_[i]]+ prob;
	                        if(residual_vnode->obs_probs[common_qnode->particleIDs_[i]] > max_prob_sum)
	                        {
	                            max_prob_sum = residual_vnode->obs_probs[common_qnode->particleIDs_[i]];
	                        }


	                }
	                child_vnode->prob_o_given_b = total_weight;
	                //std::cout << "Max prob sum " << max_prob_sum << std::endl;
	                for(int i = 0; i < common_qnode->particleIDs_.size(); i++)
	                {
	                    if(total_weight > 0) //total weight might be zero if particle weight is zero
	                    {
	                    child_vnode->particle_weights[common_qnode->particleIDs_[i]] = child_vnode->particle_weights[common_qnode->particleIDs_[i]]/total_weight;
	                    }


	                }

	                logd << " Creating node for obs " << obs << std::endl;


	        //Update upper bound lower bound
			history.Add(qnode->edge(), obs);
			if(common_qnode->default_move.value_array == NULL)
			{
				common_qnode->default_lower_bound_alpha_vector.resize(Globals::config.num_scenarios, 0);
				common_qnode->default_upper_bound_alpha_vector.resize(Globals::config.num_scenarios, 0);


				for(int i = 0; i < common_qnode->particleIDs_.size(); i++)
				{
					int scenario_id = common_qnode->particleIDs_[i];
					common_qnode->default_lower_bound_alpha_vector[scenario_id] = Hst_lb_all_a_p[ThreadID][action* NumScenarios + scenario_id].value;
					common_qnode->default_upper_bound_alpha_vector[scenario_id] = Hst_ub_all_a_p[ThreadID][action* NumScenarios + scenario_id];
				}
				int first_particle = action* NumScenarios
								+ common_qnode->particleIDs_[0];
				common_qnode->default_move = ValuedAction(
								Hst_lb_all_a_p[ThreadID][first_particle].action,0.0);
				common_qnode->default_move.value_array = (&(common_qnode->default_lower_bound_alpha_vector));
				if(Globals::config.use_sawtooth_upper_bound)
				{
					common_qnode->vnode_upper_bound_per_particle = common_qnode->default_upper_bound_alpha_vector;

				}

			}
			DESPOT::InitBounds(child_vnode, lb, ub, streams, history);
			history.RemoveLast();
	        //Init bounds using data from GPU




			logd << " New node's bounds: (" << child_vnode->lower_bound() << child_vnode->lower_bound_alpha_vector<< ", "
				<< child_vnode->upper_bound() << child_vnode->common_parent()->default_upper_bound_alpha_vector << ")" << std::endl;
	                //lower_bound += vnode->lower_bound();
			//upper_bound += vnode->upper_bound();
			//lower_bound += vnode->lower_bound()*observation_particle_size_/observation_particle_size;
			//upper_bound += vnode->upper_bound()*observation_particle_size_/observation_particle_size;
	        }


		//Scale probs
			for (std::map<OBS_TYPE, VNode*>::iterator it = children.begin();
				it != children.end(); it++) {
				VNode* child_vnode = it->second;
		                if(!child_vnode->extra_node)
		                {
		            for(int i = 0; i < common_qnode->particleIDs_.size();i++)
		            {
		                child_vnode->obs_probs[common_qnode->particleIDs_[i]] = child_vnode->obs_probs[common_qnode->particleIDs_[i]]/max_prob_sum;
		            }
		            child_vnode->prob_o_given_b = child_vnode->prob_o_given_b/max_prob_sum;
		            }
		        }

	        //Residual node
	        if(common_qnode->particleIDs_.size() > 0)
	        {
	            double total_weight = 0;
	                for(int i = 0; i < common_qnode->particleIDs_.size();i++)
	                {
	                    double prob = 1 - (residual_vnode->obs_probs[common_qnode->particleIDs_[i]]/max_prob_sum);



	                    //std::cout << "Obs Prob: res" <<  prob << " ";

			 // Terminal state is not required to be explicitly represented and may not have any observation
				residual_vnode->particle_weights[common_qnode->particleIDs_[i]] = vnode->particle_weights[common_qnode->particleIDs_[i]]* prob;
				total_weight += residual_vnode->particle_weights[common_qnode->particleIDs_[i]];

	                        residual_vnode->obs_probs[common_qnode->particleIDs_[i]] = prob;




	                }
	                residual_vnode->prob_o_given_b = total_weight;
	                for(int i = 0; i < common_qnode->particleIDs_.size(); i++)
	                {
	                    if(total_weight > 0) //total weight might be zero for residual node
	                    {
	                    residual_vnode->particle_weights[common_qnode->particleIDs_[i]] = residual_vnode->particle_weights[common_qnode->particleIDs_[i]]/total_weight;
	                    }

	                }

	                logd << " Creating node for obs " << Globals::RESIDUAL_OBS << std::endl;


			history.Add(qnode->edge(), Globals::RESIDUAL_OBS);
			//common_qnode lower bound upper bound already updated at vnodes. So not updated here
			DESPOT::InitBounds(residual_vnode, lb, ub, streams, history);
			history.RemoveLast();

			logd << " New node's bounds: (" << residual_vnode->lower_bound() << residual_vnode->lower_bound_alpha_vector<< ", "
				<< residual_vnode->upper_bound() << residual_vnode->common_parent()->default_upper_bound_alpha_vector << ")" << std::endl;
	                //lower_bound += vnode->lower_bound();

	        }


		}//Loop over actions
		if(particleIds_all_a.size() > 0)
		{
			DESPOT::PrepareGPUDataForCommonQNode(vnode->common_parent_, model, ThreadID, streams, particleIds_all_a);
		}
	}
	else
	{


	/*Expand v-node*/
	for (int action = 0; action < NumActions; action++) {
		/*Partition particles by observation*/
#ifdef RECORD_TIME
		auto start = Time::now();
#endif
		std::map<OBS_TYPE, std::vector<State*> > partitions;
		std::map<OBS_TYPE, std::vector<int> > partitions_ID;
		for (int i = 0; i < NumParticles; i++) {
			int parent_PID = particleIDs[i];
			OBS_TYPE obs;

			if(Obs_type==OBS_INT_ARRAY)
			{
				std::vector<int> tempobs;
				int* Int_obs_list = &Hst_obs_int_all_a_and_p[ThreadID][(action * NumScenarios
									+ parent_PID)*num_Obs_element_in_GPU];
				int num_obs_elements=Int_obs_list[0];
				tempobs.resize(num_obs_elements);

				for(int i=0;i<num_obs_elements;i++)
				{
					tempobs[i]=Int_obs_list[i+1];
				}

				std::hash<std::vector<int>> myhash;
				obs=myhash(tempobs);
			}
			else
			{
				obs = Hst_obs_all_a_and_p[ThreadID][action * NumScenarios
					+ parent_PID];
			}


			if (Hst_term_all_a_and_p[ThreadID][action * NumScenarios + parent_PID] == false) {
				partitions[obs].push_back(NULL);
				partitions_ID[obs].push_back(i);
			}
		}


#ifdef RECORD_TIME
		double oldValue=MakePartitionTime.load();
		MakePartitionTime.compare_exchange_weak(oldValue,oldValue+ Globals::ElapsedTime(start));
		/*Create new v-nodes for partitions, calculate the bounds*/
		auto nodestart = Time::now();
#endif

		QNode* qnode = vnode->Child(action);

		if(Globals::config.use_multi_thread_ && Globals::config.exploration_mode==UCT)
			static_cast<Shared_QNode*>(qnode)->visit_count_=1.1;

		if (partitions.size() == 0 && false) {
			cout<<"[Qnode] depth="<<vnode->depth()+1<<" obs="<< vnode->edge()<<" qnode "<<action<<" all particle termination: reward="<<Hst_r_all_a[action];
			cout<<" parent lb:"<<qnode->parent()->lower_bound()<<endl;
		} else {
		}

		double lower_bound = 0, upper_bound = 0;
		Hst_r_all_a[ThreadID][action] = Globals::Discount(vnode->depth())
				* Hst_r_all_a[ThreadID][action]
				- Globals::config.pruning_constant; //pruning_constant is used for regularization
		lower_bound = (Hst_r_all_a[ThreadID][action]);
		upper_bound = (Hst_r_all_a[ThreadID][action]);

		bool DoPrint= DESPOT::Print_nodes;
		if (FIX_SCENARIO == 1 && DoPrint) {
			cout.precision(10);
			if(action==0) cout<<endl;
			cout << "step reward (d= " << vnode->depth() + 1 << " ): "
					<< Hst_r_all_a[ThreadID][action] / (1.0f/Globals::config.num_scenarios * NumParticles)
					<< endl;
		}


		std::map<OBS_TYPE, VNode*>& children = qnode->children();
		for (std::map<OBS_TYPE, std::vector<State*> >::iterator it =
				partitions.begin(); it != partitions.end(); it++) {
			OBS_TYPE obs = it->first;
			logd << " Creating node for obs " << obs << endl;

			VNode* child_vnode;

			if (Globals::config.use_multi_thread_)
			{
				child_vnode = new Shared_VNode(partitions[obs],
						partitions_ID[obs], vnode->depth() + 1,
						static_cast<Shared_QNode*>(qnode), obs);

				if(Globals::config.exploration_mode==UCT)
					static_cast<Shared_VNode*>(child_vnode)->visit_count_=1.1;
			}
			else
				child_vnode = new VNode(partitions[obs], partitions_ID[obs],
						vnode->depth() + 1, qnode, obs);
#ifdef RECORD_TIME
			start = Time::now();
#endif

			/*Create GPU particles for the new v-node*/
			child_vnode->weight_=partitions[obs].size()/((float)NumScenarios);

			logd << " New node created!" << endl;
			children[obs] = child_vnode;

			/*Calculate initial bounds*/
			double vnode_lower_bound = 0;
			double vnode_upper_bound = 0;
			double vnode_utility_upper = 0;

			for (int i = 0; i < child_vnode->particleIDs().size(); i++) {
				int parent_PID = child_vnode->particleIDs()[i];

				vnode_lower_bound += Hst_lb_all_a_p[ThreadID][action
						* NumScenarios + parent_PID].value;
				vnode_upper_bound += Hst_ub_all_a_p[ThreadID][action
						* NumScenarios + parent_PID];				
				vnode_utility_upper += Hst_uub_all_a_p[ThreadID][action
						* NumScenarios + parent_PID];				
			}

			child_vnode->lower_bound(vnode_lower_bound);
			child_vnode->upper_bound(vnode_upper_bound-Globals::config.pruning_constant);
			child_vnode->utility_upper_bound(vnode_utility_upper);
			int first_particle = action * NumScenarios
					+ child_vnode->particleIDs()[0];
			child_vnode->default_move(
					ValuedAction(
							Hst_lb_all_a_p[ThreadID][first_particle].action,
							vnode_lower_bound));
			logd << " New node's bounds: (" << child_vnode->lower_bound()
					<< ", " << child_vnode->upper_bound() << ")" << endl;

			if (child_vnode->upper_bound() < child_vnode->lower_bound()
			// close gap because no more search can be done on leaf node
					|| child_vnode->depth() == Globals::config.search_depth - 1) {
				child_vnode->upper_bound(child_vnode->lower_bound());
			}
			
#ifdef RECORD_TIME
			init_bound_hst_t += Globals::ElapsedTime(start);
#endif

			if (FIX_SCENARIO == 1 || DoPrint) {
				cout.precision(10);
				cout << " [GPU Vnode] New node's bounds: (d= "
						<< child_vnode->depth() << " ,obs=" << obs << " , lb= "
						<< child_vnode->lower_bound() / child_vnode->weight_
						<< " ,ub= "
						<< child_vnode->upper_bound() / child_vnode->weight_
						<< " ,uub= "
						<< child_vnode->utility_upper_bound()
								/ child_vnode->weight_ << " ,weight= "
						<< child_vnode->weight_ << " )";
				if(child_vnode->Weight()==1.0/Globals::config.num_scenarios) cout<<", particle_id="<< child_vnode->particles()[0]->scenario_id;
					cout<<", WEU="<<WEU(child_vnode);
				cout  << endl;
			}

			lower_bound += child_vnode->lower_bound();
			upper_bound += child_vnode->upper_bound();

		}

#ifdef RECORD_TIME
		oldValue=CopyParticleTime.load();
		CopyParticleTime.compare_exchange_weak(oldValue,oldValue+init_bound_hst_t);
#endif
		qnode->step_reward = Hst_r_all_a[ThreadID][action];

		qnode->lower_bound(lower_bound);
		qnode->upper_bound(upper_bound);
		qnode->utility_upper_bound(
				upper_bound + Globals::config.pruning_constant);
		qnode->default_value = lower_bound; 

		qnode->Weight();
		if (FIX_SCENARIO == 1 || DoPrint) {
			cout.precision(10);
			cout << " [GPU Qnode] New qnode's bounds: (d= " << vnode->depth() + 1
					<< " ,action=" << action << ", lb= "
					<< qnode->lower_bound() / qnode->Weight() << " ,ub= "
					<< qnode->upper_bound() / qnode->Weight() << " ,uub= "
					<< qnode->utility_upper_bound() / qnode->Weight()
					<< " ,weight= " << qnode->Weight() << " )" << endl;
		}

#ifdef RECORD_TIME
		oldValue=MakeObsNodeTime.load();
		MakeObsNodeTime.compare_exchange_weak(oldValue,oldValue+ Global::ElapsedTime(nodestart) - init_bound_hst_t);
#endif

	}
	}

	if(Globals::config.use_multi_thread_)
		static_cast<Shared_VNode*>(vnode)->is_waiting_=false;

	double oldValue=TotalExpansionTime.load();
	TotalExpansionTime.compare_exchange_weak(oldValue,oldValue+ Globals::ElapsedTime(start_total));
}

int DESPOT::CalSharedMemSize() {
	int Shared_mem_per_particle;

	if (Obs_type == OBS_INT_ARRAY)
		Shared_mem_per_particle = 200;
	else
		Shared_mem_per_particle = 60;

	return Shared_mem_per_particle;
}

void DESPOT::ReadBackData(int ThreadID) {
	if (Globals::config.use_multi_thread_) {
		HANDLE_ERROR(
				hipMemcpyAsync(Hst_MC_Data[ThreadID], Dvc_MC_Data[ThreadID],
						MC_DataSize, hipMemcpyDeviceToHost,
						Globals::GetThreadCUDAStream(ThreadID)));

		logd << "Hst_MC_Data[ThreadID]" << Hst_MC_Data[ThreadID] <<"Dvc_MC_Data[ThreadID]" << Dvc_MC_Data[ThreadID] << endl;
		HANDLE_ERROR(hipStreamSynchronize(Globals::GetThreadCUDAStream(ThreadID)));
	} else {
		HANDLE_ERROR(
				hipMemcpy(Hst_MC_Data[ThreadID], Dvc_MC_Data[ThreadID],
						MC_DataSize, hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipDeviceSynchronize());
	}
}


void DESPOT::GPU_InitBounds(VNode* vnode, ScenarioLowerBound* lower_bound,
		ScenarioUpperBound* upper_bound,const DSPOMDP* model, RandomStreams& streams,
		History& history) {

	int ThreadID = 0;

	PrepareGPUDataForNode(vnode, model, ThreadID, streams);

	int blocky;
	dim3 GridDim;
	dim3 ThreadDim;
	int NumScenarios = Globals::config.num_scenarios;
	int NumParticles=vnode->num_GPU_particles_;

	int ParalllelisminStep = model->ParallelismInStep();

	int Shared_mem_per_particle = CalSharedMemSize();
	int threadx = 32;

	blocky =(NumParticles % threadx == 0) ?
			NumParticles / threadx : NumParticles / threadx + 1;
	GridDim.x = 1;
	GridDim.y = blocky;
	ThreadDim.x = threadx;
	ThreadDim.y = model->ParallelismInStep();

	if(Obs_type==OBS_INT_ARRAY)
	{

		logd<<__FUNCTION__<<": "<<__LINE__<<endl;
		logd<<__FUNCTION__<<": GridDim="<<GridDim.x<<","<<GridDim.y<<endl;
		logd<<__FUNCTION__<<": ThreadDim="<<ThreadDim.x<<","<<ThreadDim.y<<endl;
		logd<<__FUNCTION__<<": NumParticles="<<NumParticles<<endl;
		logd<<__FUNCTION__<<": vnode->GetGPUparticles()="<<vnode->GetGPUparticles()<<endl;
		logd<<__FUNCTION__<<": vnode->depth()="<<vnode->depth()<<endl;
		logd<<__FUNCTION__<<": history.Size()="<<history.Size()<<endl;
		logd<<__FUNCTION__<<": Shared_mem_per_particle="<<Shared_mem_per_particle<<endl;
		PrintThreadData(ThreadID);

		if (Globals::config.use_multi_thread_)
			_InitBounds_IntArrayObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int),
					Globals::GetThreadCUDAStream(ThreadID)>>>(Globals::config.num_scenarios,
					NumParticles, vnode->GetGPUparticles(),
					Dvc_particleIDs_long[ThreadID], Dvc_ub_all_a_p[ThreadID],
					Dvc_uub_all_a_p[ThreadID], Dvc_lb_all_a_p[ThreadID],
					Dvc_obs_all_a_and_p[ThreadID], Dvc_streams[ThreadID],
					Dvc_history[ThreadID], vnode->depth(),
					history.Size(),Shared_mem_per_particle);
		else
			_InitBounds_IntArrayObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int)>>>(
					Globals::config.num_scenarios, NumParticles,
					vnode->GetGPUparticles(),
					Dvc_particleIDs_long[ThreadID], Dvc_ub_all_a_p[ThreadID],
					Dvc_uub_all_a_p[ThreadID], Dvc_lb_all_a_p[ThreadID],
					Dvc_obs_all_a_and_p[ThreadID], Dvc_streams[ThreadID],
					Dvc_history[ThreadID], 
					vnode->depth(),
					history.Size(),Shared_mem_per_particle);


	}
	else
	{
		if (Globals::config.use_multi_thread_)
			_InitBounds_LongObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int),
					Globals::GetThreadCUDAStream(ThreadID)>>>(Globals::config.num_scenarios,
					NumParticles, vnode->GetGPUparticles(),
					Dvc_particleIDs_long[ThreadID], Dvc_ub_all_a_p[ThreadID],
					Dvc_uub_all_a_p[ThreadID], Dvc_lb_all_a_p[ThreadID],
					Dvc_obs_all_a_and_p[ThreadID], Dvc_streams[ThreadID],
					Dvc_history[ThreadID], vnode->depth(),
					history.Size());
		else
			_InitBounds_LongObs<<<GridDim, ThreadDim, threadx * Shared_mem_per_particle * sizeof(int)>>>(
					Globals::config.num_scenarios, NumParticles,
					vnode->GetGPUparticles(),
					Dvc_particleIDs_long[ThreadID], Dvc_ub_all_a_p[ThreadID],
					Dvc_uub_all_a_p[ThreadID], Dvc_lb_all_a_p[ThreadID],
					Dvc_obs_all_a_and_p[ThreadID], Dvc_streams[ThreadID],
					Dvc_history[ThreadID], vnode->depth(),
					history.Size());
	}
	ReadBackData(ThreadID);

	double vnode_lower_bound = 0;
	double vnode_upper_bound = 0;
	double vnode_utility_upper = 0;

	if(Globals::config.track_alpha_vector)
	{
		QNode* common_parent = vnode->common_parent();
		common_parent->default_lower_bound_alpha_vector.resize(Globals::config.num_scenarios, 0);
		common_parent->default_upper_bound_alpha_vector.resize(Globals::config.num_scenarios, 0);

		//This function is only called for root. So assuming particleIds are synchronized with scenarioId
		for(int i = 0; i < Globals::config.num_scenarios; i++)
		{
			common_parent->default_lower_bound_alpha_vector[i] = Hst_lb_all_a_p[ThreadID][0* NumScenarios + i].value;
			common_parent->default_upper_bound_alpha_vector[i] = Hst_ub_all_a_p[ThreadID][0* NumScenarios + i];
		}
		int first_particle = 0 * NumScenarios
						+ vnode->particleIDs()[0];
				common_parent->default_move = ValuedAction(
						Hst_lb_all_a_p[ThreadID][first_particle].action,0.0);
		common_parent->default_move.value_array = (&(common_parent->default_lower_bound_alpha_vector));
		if(Globals::config.use_sawtooth_upper_bound)
		{
			common_parent->vnode_upper_bound_per_particle = common_parent->default_upper_bound_alpha_vector;

		}

	}
	else
	{
		for (int i = 0; i < vnode->particleIDs().size(); i++) {
			int parent_PID = vnode->particleIDs()[i];

			vnode_lower_bound += Hst_lb_all_a_p[ThreadID][0
					* NumScenarios + parent_PID].value;
			vnode_upper_bound += Hst_ub_all_a_p[ThreadID][0
					* NumScenarios + parent_PID];
			vnode_utility_upper += Hst_uub_all_a_p[ThreadID][0
					* NumScenarios + parent_PID];
		}

		vnode->lower_bound(vnode_lower_bound);
		vnode->upper_bound(vnode_upper_bound-Globals::config.pruning_constant);
		vnode->utility_upper_bound(vnode_utility_upper);
		int first_particle = 0 * NumScenarios
				+ vnode->particleIDs()[0];
		vnode->default_move(ValuedAction(
				Hst_lb_all_a_p[ThreadID][first_particle].action,
				vnode_lower_bound));

		if (vnode->upper_bound() < vnode->lower_bound()
		// close gap because no more search can be done on leaf node
				|| vnode->depth() == Globals::config.search_depth - 1) {
			vnode->upper_bound(vnode->lower_bound());
		}
	}
}


void DESPOT::GPU_UpdateParticles(VNode* vnode, ScenarioLowerBound* lb,
		ScenarioUpperBound* ub, const DSPOMDP* model, RandomStreams& streams,
		History& history) {

	int ThreadID = 0;
	if (Globals::config.use_multi_thread_)
		ThreadID = Globals::MapThread(this_thread::get_id());

	int NumActions = model->NumActions();
	int NumObs = model->NumObservations();

	if(Globals::config.use_multi_thread_)
		static_cast<Shared_VNode*>(vnode)->is_waiting_=true;

	auto start_total = Time::now();

	/*Update streams, history, and particles into GPU*/
	PrepareGPUDataForNode(vnode, model, ThreadID, streams);


	if(!Globals::config.track_alpha_vector)
	{
		/*get the GPU particles of the parent v-node*/

		MCSimulation(vnode, ThreadID,model, streams,history,false);
	}
}


void DESPOT::PrepareGPUDataForRoot(VNode* node, const DSPOMDP* model,
		const std::vector<int>& particleIDs, std::vector<State*>& particles
		) {

	int NumParticles = particleIDs.size();

	/* Root particles are managed memory that can easily copy over CPU data */
	Dvc_State* new_particles = model->AllocGPUParticles(NumParticles, MEMORY_MODE(ALLOC_ROOT));
	
	model->CopyParticlesToGPU(new_particles, particles, true);

	model->CopyParticleIDsToGPU(Dvc_particleIDs_long[0], particleIDs);

	/* This operation is to reset the Dvc_streams info (reset pos to 0) */
	model->CopyGPUParticlesFromParent(new_particles, new_particles, 0,
			Dvc_particleIDs_long[0], particles.size(),
			Dvc_streams[0], 0,
			false);

	node->weight_ =NumParticles/((float)Globals::config.num_scenarios);
	node->AssignGPUparticles(new_particles, particles.size());
}

void DESPOT::PrintGPUData(int num_searches) {
	cout.precision(5);
	if (Globals::config.use_multi_thread_)
		cout << "ExpansionCount (total/per-search)=" << Globals::CountExpanded() << "/"
				<< Globals::CountExpanded() / num_searches << endl;
	else
		cout << "ExpansionCount (total/per-search)=" << Globals::CountExpanded() << "/"
				<< Globals::CountExpanded() / num_searches << endl;
	cout.precision(3);
}




void PrintThreadData(int ThreadID){
	logd<<__FUNCTION__<<": ThreadID="<<ThreadID<<endl;
	logd<<__FUNCTION__<<": Dvc_particleIDs_long[ThreadID]="<<Dvc_particleIDs_long[ThreadID]<<endl;
	logd<<__FUNCTION__<<": Dvc_ub_all_a_p[ThreadID]="<<Dvc_ub_all_a_p[ThreadID]<<endl;
	logd<<__FUNCTION__<<": Dvc_uub_all_a_p[ThreadID]="<<Dvc_uub_all_a_p[ThreadID]<<endl;
	logd<<__FUNCTION__<<": Dvc_lb_all_a_p[ThreadID]="<<Dvc_lb_all_a_p[ThreadID]<<endl;
	logd<<__FUNCTION__<<": Dvc_streams[ThreadID]="<<Dvc_streams[ThreadID]<<endl;
	logd<<__FUNCTION__<<": Dvc_streams[ThreadID]->position_="<<Dvc_streams[ThreadID]->position_<<endl;
	logd<<__FUNCTION__<<": Dvc_streams[ThreadID]->streams_="<<Dvc_streams[ThreadID]->streams_<<endl;
	logd<<__FUNCTION__<<": Dvc_history[ThreadID]="<<Dvc_history[ThreadID]<<endl;
	logd<<__FUNCTION__<<": Dvc_history[ThreadID]->actions_="<<Dvc_history[ThreadID]->actions_<<endl;

}


__global__ void FreeHistory(Dvc_History* history, int num_particles) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < num_particles) {
		history[i].currentSize_ = 0;
	}
}

void initGPUHistory() {
	int thread_count = 1;
	if (Globals::config.use_multi_thread_)
		thread_count = Globals::config.NUM_THREADS;

	Dvc_history = new Dvc_History*[thread_count];
	for (int i = 0; i < thread_count; i++) {
		HANDLE_ERROR(
				hipMallocManaged((void** )&Dvc_history[i],
						1 * sizeof(Dvc_History)));
	}
	Dvc_history[0]->CreateMemoryPool(0);
	cout<<"Globals::config.search_depth="<<Globals::config.search_depth<<endl;
	for (int i = 0; i < thread_count; i++) {
		if (Globals::config.use_multi_thread_)
			Dvc_History::InitInGPU(Globals::config.num_scenarios, Dvc_history[i],
					Globals::config.search_depth);
		else
			Dvc_History::InitInGPU(Globals::config.num_scenarios, Dvc_history[i],
					Globals::config.search_depth);
	}
	HANDLE_ERROR(hipDeviceSynchronize());
}

void clearGPUHistory() {
	int thread_count = 1;
	if (Globals::config.use_multi_thread_)
		thread_count = Globals::config.NUM_THREADS;
	for (int i = 0; i < thread_count; i++) {
		if (Dvc_history[i] != NULL) {
			dim3 grid((Globals::config.num_scenarios + MC_DIM - 1) / MC_DIM, 1);
			dim3 threads(MC_DIM, 1);
			FreeHistory<<<1, 1,1>>>(Dvc_history[i], Globals::config.num_scenarios);
			HANDLE_ERROR(hipDeviceSynchronize());
		}
	}
	Dvc_history[0]->DestroyMemoryPool(0);

	if (Dvc_history)
	{
		delete[] Dvc_history;
		Dvc_history = NULL;
	}
}

} // namespace despot
